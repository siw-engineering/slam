#include "hip/hip_runtime.h"
/*
 * This file is part of ElasticFusion.
 *
 * Copyright (C) 2015 Imperial College London
 * 
 * The use of the code within this file and all code within files that 
 * make up the software that is ElasticFusion is permitted for 
 * non-commercial purposes only.  The full terms and conditions that 
 * apply to the code within this file are detailed within the LICENSE.txt 
 * file and at <http://www.imperial.ac.uk/dyson-robotics-lab/downloads/elastic-fusion/elastic-fusion-license/> 
 * unless explicitly stated.  By downloading this file you agree to 
 * comply with these terms.
 *
 * If you wish to use any of this code for commercial purposes then 
 * please email researchcontracts.engineering@imperial.ac.uk.
 *
 * Software License Agreement (BSD License)
 *
 *  Copyright (c) 2011, Willow Garage, Inc.
 *  All rights reserved.
 *
 *  Redistribution and use in source and binary forms, with or without
 *  modification, are permitted provided that the following conditions
 *  are met:
 *
 *   * Redistributions of source code must retain the above copyright
 *     notice, this list of conditions and the following disclaimer.
 *   * Redistributions in binary form must reproduce the above
 *     copyright notice, this list of conditions and the following
 *     disclaimer in the documentation and/or other materials provided
 *     with the distribution.
 *   * Neither the name of Willow Garage, Inc. nor the names of its
 *     contributors may be used to endorse or promote products derived
 *     from this software without specific prior written permission.
 *
 *  THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS
 *  "AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT
 *  LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS
 *  FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE
 *  COPYRIGHT OWNER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT,
 *  INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING,
 *  BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 *  LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 *  CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT
 *  LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN
 *  ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
 *  POSSIBILITY OF SUCH DAMAGE.
 *
 *  Author: Anatoly Baskeheev, Itseez Ltd, (myname.mysurname@mycompany.com)
 */

#include "cudafuncs.cuh"
#include "convenience.cuh"
#include "operators.cuh"

__global__ void pyrDownGaussKernel (const PtrStepSz<float> src, PtrStepSz<float> dst, float sigma_color)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= dst.cols || y >= dst.rows)
        return;

    const int D = 5;

    int center = src.ptr (2 * y)[2 * x];

    int x_mi = max(0, 2*x - D/2) - 2*x;
    int y_mi = max(0, 2*y - D/2) - 2*y;

    int x_ma = min(src.cols, 2*x -D/2+D) - 2*x;
    int y_ma = min(src.rows, 2*y -D/2+D) - 2*y;

    float sum = 0;
    float wall = 0;

    float weights[] = {0.375f, 0.25f, 0.0625f} ;

    for(int yi = y_mi; yi < y_ma; ++yi)
        for(int xi = x_mi; xi < x_ma; ++xi)
        {
            int val = src.ptr (2*y + yi)[2*x + xi];

            if (abs (val - center) < 3 * sigma_color)
            {
                sum += val * weights[abs(xi)] * weights[abs(yi)];
                wall += weights[abs(xi)] * weights[abs(yi)];
            }
        }


    dst.ptr (y)[x] = static_cast<int>(sum / wall);
}

void pyrDown(const DeviceArray2D<unsigned short> & src, DeviceArray2D<unsigned short> & dst)
{
    dst.create (src.rows () / 2, src.cols () / 2);

    dim3 block (32, 8);
    dim3 grid (getGridDim (dst.cols (), block.x), getGridDim (dst.rows (), block.y));

    const float sigma_color = 30;

    pyrDownGaussKernel<<<grid, block>>>(src, dst, sigma_color);
    cudaCheckError();
}

__device__ float3 projectPoint(float3 p, int rows, int cols, float cx, float cy, float fx, float fy, float maxDepth)
{
    float3 pt = make_float3(
                ((((fx * p.x) / p.z) + cx) - (cols * 0.5)) / (cols * 0.5),
                ((((fy * p.y) / p.z) + cy) - (rows * 0.5)) / (rows * 0.5),
                p.z / maxDepth
                );
    return pt;
}

// Generate a vertex map 'vmap' based on the depth map 'depth' and camera parameters
__global__ void computeVmapKernel(const PtrStepSz<float> depth, PtrStep<float> vmap, float fx_inv, float fy_inv, float cx, float cy, float depthCutoff)
{
    int u = threadIdx.x + blockIdx.x * blockDim.x;
    int v = threadIdx.y + blockIdx.y * blockDim.y;

    if(u < depth.cols && v < depth.rows)
    {
        float z = depth.ptr(v)[u] /*/ 1000.f*/; // load and convert: mm -> meters

        if(z != 0 && z < depthCutoff /*&& m == maskID*/) //FIXME
        {
            float vx = z * (u - cx) * fx_inv;
            float vy = z * (v - cy) * fy_inv;
            float vz = z;

            vmap.ptr (v                 )[u] = vx;
            vmap.ptr (v + depth.rows    )[u] = vy;
            vmap.ptr (v + depth.rows * 2)[u] = vz;

            float3 p = make_float3(vx, vy, vz);
            p = projectPoint(p, depth.rows, depth.cols, cx, cy, 1/fx_inv, 1/fy_inv, depthCutoff);
            // printf(" p.x = %f\n", p.x);

        }
        else
        {
            vmap.ptr (v)[u] = __int_as_float(0x7fffffff); /*HIP_NAN_F*/
        }
    }
}


void createVMap(const CameraModel& intr, const DeviceArray2D<float> & depth, DeviceArray2D<float> & vmap, const float depthCutoff)
{
    vmap.create (depth.rows () * 3, depth.cols ());

    dim3 block (32, 8);
    dim3 grid (1, 1, 1);
    grid.x = getGridDim (depth.cols (), block.x);
    grid.y = getGridDim (depth.rows (), block.y);

    float fx = intr.fx, cx = intr.cx;
    float fy = intr.fy, cy = intr.cy;

    computeVmapKernel<<<grid, block>>>(depth, vmap, 1.f / fx, 1.f / fy, cx, cy, depthCutoff);
    cudaSafeCall(hipGetLastError());
}

__device__ float getRadius(float fx, float fy, float depth, float norm_z)
{
    float meanFocal = ((1.0 / abs(fx)) + (1.0 / abs(fy))) / 2.0;
    
    const float sqrt2 = 1.41421356237f;
    
    float radius = (depth / meanFocal) * sqrt2;

    float radius_n = radius;

    radius_n = radius_n / abs(norm_z);

    radius_n = min(2.0f * radius, radius_n);

    return radius_n;
}

__device__ float3 getNormal(const PtrStepSz<float> depth, float3 vpos, float cx, float cy, float fx, float fy, int u, int v, int rows, int cols)
{
    // if (!isnan (v00.x) && !isnan (v01.x) && !isnan (v10.x)) TO DO check for nan

    float z1 = depth.ptr(v)[u + 1];
    float z2 = depth.ptr(v)[u - 1];
    float z3 = depth.ptr(v + 1)[u];
    float z4 = depth.ptr(v - 1)[u];

    float3 v1 = make_float3(z1 * (u + 1 - cx) / fx,  z1 * (v - cy) / fy,  z1); 
    float3 v2 = make_float3(z2 * (u - 1 - cx) / fx,  z2 * (v - cy) / fy,  z2); 
    float3 v3 = make_float3(z3 * (u - cx) / fx,  z3 * (v + 1 - cy) / fy,  z3); 
    float3 v4 = make_float3(z4 * (u - cx) / fx,  z4 * (v - 1- cy) / fy,  z4); 

    if (!isnan (v1.x) && !isnan (v2.x) && !isnan (v3.x))
    {

        float3 del_x = make_float3(((vpos.x + v2.x)/2 - (vpos.x + v1.x)/2), ((vpos.y + v2.y)/2 - (vpos.y + v1.y)/2), ((vpos.z + v2.z)/2 - (vpos.z + v1.z)/2));
        float3 del_y = make_float3(((vpos.x + v4.x)/2 - (vpos.x + v3.x)/2), ((vpos.y + v4.y)/2 - (vpos.y + v3.y)/2), ((vpos.z + v4.z)/2 - (vpos.z + v3.z)/2));

        return normalized(cross(del_x, del_y));    
    }
    else
        return make_float3(0,0,0); // TO DO not sure if putting 0s is right
}


__device__ float encodeColor(float3 c)
{
    int rgb = 0;
    rgb = int(round(c.x * 255.0f));
    rgb = (rgb << 8) + int(round(c.y * 255.0f));
    rgb = (rgb << 8) + int(round(c.z * 255.0f));
    return  (float)rgb;
}

__device__ float3 decodeColor(float c)
{
    float3 col;
    col.x = float(int(c) >> 16 & 0xFF) / 255.0f;
    col.y = float(int(c) >> 8 & 0xFF) / 255.0f;
    col.z = float(int(c) & 0xFF) / 255.0f;
    return col;
}


__device__ float confidence(float cx, float cy, float x, float y, float weighting)
{
    const float maxRadDist = 400; //sqrt((width * 0.5)^2 + (height * 0.5)^2)
    const float twoSigmaSquared = 0.72; //2*(0.6^2) from paper

    float3 pixelPosCentered = make_float3(x-cx, y-cy, 0);
    // vec2 pixelPosCentered = vec2(x, y) - cam.xy;
    float radialDist = sqrt(dot(pixelPosCentered, pixelPosCentered)) / maxRadDist;
    return exp((-(radialDist * radialDist) / twoSigmaSquared)) * weighting;
}


__global__ void initModelBufferKernel(float cx, float cy, float fx, float fy, int rows, int cols, float max_depth, float* model_buffer,  int* count, const PtrStepSz<float> vmap, const PtrStepSz<float> nmap, const float* rgb)
{
    int u = threadIdx.x + blockIdx.x * blockDim.x;
    int v = threadIdx.y + blockIdx.y * blockDim.y;


    float vz = vmap.ptr(v + rows*2)[u];
    atomicAdd(count, 1);

    if ((vz < 0) || (vz > max_depth))
    {
        return;
    }



    // replace this, hardcoding temporarily
    int rows_mb, cols_mb;
    rows_mb = cols_mb = 3072;

    int i = cols*v + u;
    // int i = *count;
    //writing vertex and confidence
    model_buffer[i] = vmap.ptr(v)[u];
    model_buffer[i+ rows_mb*cols_mb] = vmap.ptr(v + rows)[u];
    model_buffer[i+2*rows_mb*cols_mb] = vz;
    model_buffer[i+3*rows_mb*cols_mb] = confidence(cx, cy, u, v, 1);

    // color encoding
    float3 c;
    float ec ;
    c.x = rgb[v*cols*3 + u*3 + 0];
    c.y = rgb[v*cols*3 + u*3 + 1];
    c.z = rgb[v*cols*3 + u*3 + 2];
    ec = encodeColor(c);

    // //writing color and time
    // model_buffer[i+4*rows_mb*cols_mb] = ec; //x // TO DO UPDATE disabling color encoding, don't know what will happen
    // model_buffer[i+5*rows_mb*cols_mb] = 0;//y
    // model_buffer[i+6*rows_mb*cols_mb] = 1;//z
    // model_buffer[i+7*rows_mb*cols_mb] = 1;//w time
    model_buffer[i+4*rows_mb*cols_mb] = c.x; //x
    model_buffer[i+5*rows_mb*cols_mb] = c.y;//y
    model_buffer[i+6*rows_mb*cols_mb] = c.z;//z
    model_buffer[i+7*rows_mb*cols_mb] = 1;//w time

    //writing normals
    model_buffer[i+8*rows_mb*cols_mb] = nmap.ptr(v)[u];
    model_buffer[i+9*rows_mb*cols_mb] = nmap.ptr(v + rows)[u];
    model_buffer[i+10*rows_mb*cols_mb] = nmap.ptr(v + rows*2)[u];
    model_buffer[i+11*rows_mb*cols_mb] = getRadius(fx, fy, vmap.ptr(v + rows*2)[u], nmap.ptr(v + rows*2)[u]);

}

void initModelBuffer(const CameraModel& intr, const float depthCutOff, DeviceArray<float> & model_buffer, int* h_count, const DeviceArray2D<float> & vmap, const DeviceArray2D<float> & nmap, const DeviceArray<float> & rgb )
{
    int *d_count;
    hipMalloc((void**)&d_count, sizeof(int));
    hipMemcpy(d_count, h_count, sizeof(int), hipMemcpyHostToDevice);

    int cols, rows;
    rows = vmap.rows()/3;
    cols = vmap.cols();
    dim3 block (32, 8);
    dim3 grid (1, 1, 1);
    grid.x = getGridDim (cols, block.x);
    grid.y = getGridDim (rows, block.y);

    initModelBufferKernel<<<grid, block>>>(intr.cx, intr.cy, intr.fx, intr.fy, rows, cols, depthCutOff, model_buffer, d_count, vmap, nmap, rgb);
    hipDeviceSynchronize();
    cudaCheckError();
    hipMemcpy(h_count, d_count, sizeof(int), hipMemcpyDeviceToHost);
}

// __global__ void kernelCodeKernel(float *result)
// {
//     int index = threadIdx.x+blockIdx.x*blockDim.x;
//     atomicAdd(result, 1.0f);
    
// }
// void kernelCode(){

//     float h_result, *d_result;
//     hipMalloc((void **)&d_result, sizeof(float));
//     h_result = 0.0f;
//     hipMemcpy(d_result, &h_result, sizeof(float), hipMemcpyHostToDevice);

//     int rows, cols;
//     rows = 640;
//     cols = 480;
//     dim3 block(32, 8);
//     dim3 grid(getGridDim(cols, block.x), getGridDim(rows, block.y));

//     kernelCodeKernel<<<grid, block>>>(d_result);
//     hipDeviceSynchronize();
//     hipMemcpy(&h_result, d_result, sizeof(float), hipMemcpyDeviceToHost);
//     std::cout<< "result = " << h_result << std::endl;
// }

__global__ void computeNmapKernel(int rows, int cols, const PtrStep<float> vmap, PtrStep<float> nmap)
{
    int u = threadIdx.x + blockIdx.x * blockDim.x;
    int v = threadIdx.y + blockIdx.y * blockDim.y;

    if (u >= cols || v >= rows)
        return;

    if (u == cols - 1 || v == rows - 1)
    {
        nmap.ptr (v)[u] = __int_as_float(0x7fffffff); /*HIP_NAN_F*/
        return;
    }

    float3 v00, v01, v10;
    v00.x = vmap.ptr (v  )[u];
    v01.x = vmap.ptr (v  )[u + 1];
    v10.x = vmap.ptr (v + 1)[u];

    if (!isnan (v00.x) && !isnan (v01.x) && !isnan (v10.x))
    {
        v00.y = vmap.ptr (v + rows)[u];
        v01.y = vmap.ptr (v + rows)[u + 1];
        v10.y = vmap.ptr (v + 1 + rows)[u];

        v00.z = vmap.ptr (v + 2 * rows)[u];
        v01.z = vmap.ptr (v + 2 * rows)[u + 1];
        v10.z = vmap.ptr (v + 1 + 2 * rows)[u];

        float3 r = normalized (cross (v01 - v00, v10 - v00));

        nmap.ptr (v       )[u] = r.x;
        nmap.ptr (v + rows)[u] = r.y;
        nmap.ptr (v + 2 * rows)[u] = r.z;
    }
    else
        nmap.ptr (v)[u] = __int_as_float(0x7fffffff); /*HIP_NAN_F*/
}

void createNMap(const DeviceArray2D<float>& vmap, DeviceArray2D<float>& nmap)
{
    nmap.create (vmap.rows (), vmap.cols ());

    int rows = vmap.rows () / 3;
    int cols = vmap.cols ();

    dim3 block (32, 8);
    dim3 grid (1, 1, 1);
    grid.x = getGridDim (cols, block.x);
    grid.y = getGridDim (rows, block.y);

    computeNmapKernel<<<grid, block>>>(rows, cols, vmap, nmap);
    cudaSafeCall (hipGetLastError ());
}

__global__ void tranformMapsKernel(int rows, int cols, const PtrStep<float> vmap_src, const PtrStep<float> nmap_src,
                                   const mat33 Rmat, const float3 tvec, PtrStepSz<float> vmap_dst, PtrStep<float> nmap_dst)
{
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;

    if (x < cols && y < rows)
    {
        //vertexes
        float3 vsrc, vdst = make_float3 (__int_as_float(0x7fffffff), __int_as_float(0x7fffffff), __int_as_float(0x7fffffff));
        vsrc.x = vmap_src.ptr (y)[x];

        if (!isnan (vsrc.x))
        {
            vsrc.y = vmap_src.ptr (y + rows)[x];
            vsrc.z = vmap_src.ptr (y + 2 * rows)[x];

            vdst = Rmat * vsrc + tvec;

            vmap_dst.ptr (y + rows)[x] = vdst.y;
            vmap_dst.ptr (y + 2 * rows)[x] = vdst.z;
        }

        vmap_dst.ptr (y)[x] = vdst.x;

        //normals
        float3 nsrc, ndst = make_float3 (__int_as_float(0x7fffffff), __int_as_float(0x7fffffff), __int_as_float(0x7fffffff));
        nsrc.x = nmap_src.ptr (y)[x];

        if (!isnan (nsrc.x))
        {
            nsrc.y = nmap_src.ptr (y + rows)[x];
            nsrc.z = nmap_src.ptr (y + 2 * rows)[x];

            ndst = Rmat * nsrc;

            nmap_dst.ptr (y + rows)[x] = ndst.y;
            nmap_dst.ptr (y + 2 * rows)[x] = ndst.z;
        }

        nmap_dst.ptr (y)[x] = ndst.x;
    }
}

void tranformMaps(const DeviceArray2D<float>& vmap_src,
                  const DeviceArray2D<float>& nmap_src,
                  const mat33& Rmat, const float3& tvec,
                  DeviceArray2D<float>& vmap_dst, DeviceArray2D<float>& nmap_dst)
{
    int cols = vmap_src.cols();
    int rows = vmap_src.rows() / 3;

    vmap_dst.create(rows * 3, cols);
    nmap_dst.create(rows * 3, cols);

    dim3 block(32, 8);
    dim3 grid(1, 1, 1);
    grid.x = getGridDim(cols, block.x);
    grid.y = getGridDim(rows, block.y);

    tranformMapsKernel<<<grid, block>>>(rows, cols, vmap_src, nmap_src, Rmat, tvec, vmap_dst, nmap_dst);
    cudaSafeCall(hipGetLastError());
}

__global__ void copyMapsKernel(int rows, int cols, const float * vmap_src, const float * nmap_src,
                               PtrStepSz<float> vmap_dst, PtrStep<float> nmap_dst)
{
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;

    if (x < cols && y < rows)
    {
        //vertexes
        float3 vsrc, vdst = make_float3 (__int_as_float(0x7fffffff), __int_as_float(0x7fffffff), __int_as_float(0x7fffffff));

        vsrc.x = vmap_src[y * cols * 4 + (x * 4) + 0];
        vsrc.y = vmap_src[y * cols * 4 + (x * 4) + 1];
        vsrc.z = vmap_src[y * cols * 4 + (x * 4) + 2];

        if(!(vsrc.z == 0))
        {
            vdst = vsrc;
        }

        vmap_dst.ptr (y)[x] = vdst.x;
        vmap_dst.ptr (y + rows)[x] = vdst.y;
        vmap_dst.ptr (y + 2 * rows)[x] = vdst.z;

        //normals
        float3 nsrc, ndst = make_float3 (__int_as_float(0x7fffffff), __int_as_float(0x7fffffff), __int_as_float(0x7fffffff));

        nsrc.x = nmap_src[y * cols * 4 + (x * 4) + 0];
        nsrc.y = nmap_src[y * cols * 4 + (x * 4) + 1];
        nsrc.z = nmap_src[y * cols * 4 + (x * 4) + 2];

        if(!(vsrc.z == 0))
        {
            ndst = nsrc;
        }

        nmap_dst.ptr (y)[x] = ndst.x;
        nmap_dst.ptr (y + rows)[x] = ndst.y;
        nmap_dst.ptr (y + 2 * rows)[x] = ndst.z;
    }
}

void copyMaps(const DeviceArray<float>& vmap_src,
              const DeviceArray<float>& nmap_src,
              DeviceArray2D<float>& vmap_dst,
              DeviceArray2D<float>& nmap_dst)
{
    int cols = vmap_dst.cols();
    int rows = vmap_dst.rows() / 3;

    vmap_dst.create(rows * 3, cols);
    nmap_dst.create(rows * 3, cols);

    dim3 block(32, 8);
    dim3 grid(1, 1, 1);
    grid.x = getGridDim(cols, block.x);
    grid.y = getGridDim(rows, block.y);

    copyMapsKernel<<<grid, block>>>(rows, cols, vmap_src, nmap_src, vmap_dst, nmap_dst);
    cudaSafeCall(hipGetLastError());
}



__global__ void copyMapsKernel2D_2_2D(int rows, int cols, PtrStepSz<float> vmap_src, PtrStep<float> nmap_src,
                               PtrStepSz<float> vmap_dst, PtrStep<float> nmap_dst)
{
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;

    if (x < cols && y < rows)
    {
        // vertexes
        float3 vsrc, vdst = make_float3 (__int_as_float(0x7fffffff), __int_as_float(0x7fffffff), __int_as_float(0x7fffffff));

        vsrc.x = vmap_src.ptr(y)[x];
        vsrc.y = vmap_src.ptr(y+rows)[x];
        vsrc.z = vmap_src.ptr(y+2*rows)[x];

        if(!(vsrc.z == 0))
        {
            vdst = vsrc;
        }

        vmap_dst.ptr (y)[x] = vdst.x;
        vmap_dst.ptr (y + rows)[x] = vdst.y;
        vmap_dst.ptr (y + 2 * rows)[x] = vdst.z;

        //normals
        float3 nsrc, ndst = make_float3 (__int_as_float(0x7fffffff), __int_as_float(0x7fffffff), __int_as_float(0x7fffffff));

        nsrc.x = nmap_src.ptr(y)[x];
        nsrc.y = nmap_src.ptr(y+rows)[x];
        nsrc.z = nmap_src.ptr(y+2*rows)[x];

        if(!(vsrc.z == 0))
        {
            ndst = nsrc;
        }
        nmap_dst.ptr (y)[x] = ndst.x;
        nmap_dst.ptr (y + rows)[x] = ndst.y;
        nmap_dst.ptr (y + 2 * rows)[x] = ndst.z;
    }
}

void copyMaps(const DeviceArray2D<float>& vmap_src,
              const DeviceArray2D<float>& nmap_src,
              DeviceArray2D<float>& vmap_dst,
              DeviceArray2D<float>& nmap_dst)
{
    int cols = vmap_dst.cols();
    int rows = vmap_dst.rows() / 3;

    vmap_dst.create(rows * 3, cols);
    nmap_dst.create(rows * 3, cols);

    dim3 block(32, 8);
    dim3 grid(1, 1, 1);
    grid.x = getGridDim(cols, block.x);
    grid.y = getGridDim(rows, block.y);

    copyMapsKernel2D_2_2D<<<grid, block>>>(rows, cols, vmap_src, nmap_src, vmap_dst, nmap_dst);
    cudaSafeCall(hipGetLastError());
}

__global__ void copyMapsKernel2D_2_1D(int rows, int cols, PtrStepSz<float> vmap_src, PtrStep<float> nmap_src,
                                 float * vmap_dst,   float * nmap_dst)
{
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;

    if (x < cols && y < rows)
    {
        //vertexes
        float4 vsrc, vdst = make_float4 (__int_as_float(0x7fffffff), __int_as_float(0x7fffffff), __int_as_float(0x7fffffff),  __int_as_float(0x7fffffff));

        vsrc.x = vmap_src.ptr (y)[x];
        vsrc.y = vmap_src.ptr (y + rows)[x];
        vsrc.z = vmap_src.ptr (y + 2 * rows)[x];
        vsrc.w = vmap_src.ptr (y + 3 * rows)[x];


        if(!(vsrc.z == 0))
        {
            vdst = vsrc;
        }

        vmap_dst[y * cols * 4 + (x * 4) + 0] = vdst.x;
        vmap_dst[y * cols * 4 + (x * 4) + 1] = vdst.y;
        vmap_dst[y * cols * 4 + (x * 4) + 2] = vdst.z;
        vmap_dst[y * cols * 4 + (x * 4) + 3] = vdst.w;


        //normals
        float4 nsrc, ndst = make_float4 (__int_as_float(0x7fffffff), __int_as_float(0x7fffffff), __int_as_float(0x7fffffff),  __int_as_float(0x7fffffff));

        nsrc.x = nmap_src.ptr (y)[x] ;
        nsrc.y = nmap_src.ptr (y + rows)[x] ;
        nsrc.z = nmap_src.ptr (y + 2 * rows)[x] ;
        nsrc.w = nmap_src.ptr (y + 3 * rows)[x] ;


        if(!(vsrc.z == 0))
        {
            ndst = nsrc;
        }

        nmap_dst[y * cols * 4 + (x * 4) + 0]= ndst.x;
        nmap_dst[y * cols * 4 + (x * 4) + 1]= ndst.y;
        nmap_dst[y * cols * 4 + (x * 4) + 2]= ndst.z;
        nmap_dst[y * cols * 4 + (x * 4) + 3]= ndst.w;

    }
}

void copyMaps(const DeviceArray2D<float>& vmap_src,
              const DeviceArray2D<float>& nmap_src,
              DeviceArray<float>& vmap_dst,
              DeviceArray<float>& nmap_dst)
{
    int rows = vmap_src.rows() / 4;
    int cols = vmap_src.cols();

    vmap_dst.create(rows * 4 * cols);
    nmap_dst.create(rows * 4 * cols);

    dim3 block(32, 8);
    dim3 grid(1, 1, 1);
    grid.x = getGridDim(cols, block.x);
    grid.y = getGridDim(rows, block.y);


    copyMapsKernel2D_2_1D<<<grid, block>>>(rows, cols, vmap_src, nmap_src, vmap_dst, nmap_dst);
    cudaSafeCall(hipGetLastError());
}

__global__ void copyDMapsKernel2D_2_2D(int rows, int cols, PtrStepSz<float> dmap_src, PtrStepSz<float> dmap_dst)
{
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;

    if (x < cols && y < rows)
    {
        dmap_dst.ptr(y)[x] = dmap_src.ptr(y)[x];
    }
}

void copyDMaps(const DeviceArray2D<float>& dmap_src,
              DeviceArray2D<float>& dmap_dst)
{
    int cols = dmap_src.cols();
    int rows = dmap_src.rows();


    dim3 block(32, 8);
    dim3 grid(1, 1, 1);
    grid.x = getGridDim(cols, block.x);
    grid.y = getGridDim(rows, block.y);

    copyDMapsKernel2D_2_2D<<<grid, block>>>(rows, cols, dmap_src, dmap_dst);
    cudaSafeCall(hipGetLastError());
}

__global__ void pyrDownKernelGaussF(const PtrStepSz<float> src, PtrStepSz<float> dst, float * gaussKernel)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= dst.cols || y >= dst.rows)
        return;

    const int D = 5;

    float center = src.ptr (2 * y)[2 * x];

    int tx = min (2 * x - D / 2 + D, src.cols - 1);
    int ty = min (2 * y - D / 2 + D, src.rows - 1);
    int cy = max (0, 2 * y - D / 2);

    float sum = 0;
    int count = 0;

    for (; cy < ty; ++cy)
    {
        for (int cx = max (0, 2 * x - D / 2); cx < tx; ++cx)
        {
            if(!isnan(src.ptr (cy)[cx]))
            {
                sum += src.ptr (cy)[cx] * gaussKernel[(ty - cy - 1) * 5 + (tx - cx - 1)];
                count += gaussKernel[(ty - cy - 1) * 5 + (tx - cx - 1)];
            }
        }
    }
    dst.ptr (y)[x] = (float)(sum / (float)count);
}

template<bool normalize>
__global__ void resizeMapKernel(int drows, int dcols, int srows, const PtrStep<float> input, PtrStep<float> output)
{
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;

    if (x >= dcols || y >= drows)
        return;

    const float qnan = __int_as_float(0x7fffffff);

    int xs = x * 2;
    int ys = y * 2;

    float x00 = input.ptr (ys + 0)[xs + 0];
    float x01 = input.ptr (ys + 0)[xs + 1];
    float x10 = input.ptr (ys + 1)[xs + 0];
    float x11 = input.ptr (ys + 1)[xs + 1];

    if (isnan (x00) || isnan (x01) || isnan (x10) || isnan (x11))
    {
        output.ptr (y)[x] = qnan;
        return;
    }
    else
    {
        float3 n;

        n.x = (x00 + x01 + x10 + x11) / 4;

        float y00 = input.ptr (ys + srows + 0)[xs + 0];
        float y01 = input.ptr (ys + srows + 0)[xs + 1];
        float y10 = input.ptr (ys + srows + 1)[xs + 0];
        float y11 = input.ptr (ys + srows + 1)[xs + 1];

        n.y = (y00 + y01 + y10 + y11) / 4;

        float z00 = input.ptr (ys + 2 * srows + 0)[xs + 0];
        float z01 = input.ptr (ys + 2 * srows + 0)[xs + 1];
        float z10 = input.ptr (ys + 2 * srows + 1)[xs + 0];
        float z11 = input.ptr (ys + 2 * srows + 1)[xs + 1];

        n.z = (z00 + z01 + z10 + z11) / 4;

        if (normalize)
            n = normalized (n);

        output.ptr (y        )[x] = n.x;
        output.ptr (y + drows)[x] = n.y;
        output.ptr (y + 2 * drows)[x] = n.z;
    }
}

template<bool normalize>
void resizeMap(const DeviceArray2D<float>& input, DeviceArray2D<float>& output)
{
    int in_cols = input.cols ();
    int in_rows = input.rows () / 3;

    int out_cols = in_cols / 2;
    int out_rows = in_rows / 2;

    output.create (out_rows * 3, out_cols);

    dim3 block (32, 8);
    dim3 grid (getGridDim (out_cols, block.x), getGridDim (out_rows, block.y));
    resizeMapKernel<normalize><< < grid, block>>>(out_rows, out_cols, in_rows, input, output);
    cudaCheckError();
    cudaSafeCall (hipDeviceSynchronize ());
}

void resizeVMap(const DeviceArray2D<float>& input, DeviceArray2D<float>& output)
{
    resizeMap<false>(input, output);
}

void resizeNMap(const DeviceArray2D<float>& input, DeviceArray2D<float>& output)
{
    resizeMap<true>(input, output);
}

//FIXME Remove
/*
void launch_kernel(float4 *pos, unsigned int mesh_width,
                   unsigned int mesh_height, float time)
{
    // execute the kernel
    dim3 block(8, 8, 1);
    dim3 grid(mesh_width / block.x, mesh_height / block.y, 1);
    simple_vbo_kernel<<< grid, block>>>(pos, mesh_width, mesh_height, time);
}*/

//FIXME Remove
/*
__global__ void testKernel(hipSurfaceObject_t tex)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= 960 || y >= 540)
        return;

    / *
    const int D = 5;

    float center = src.ptr (2 * y)[2 * x];

    int tx = min (2 * x - D / 2 + D, src.cols - 1);
    int ty = min (2 * y - D / 2 + D, src.rows - 1);
    int cy = max (0, 2 * y - D / 2);

    float sum = 0;
    int count = 0;

    for (; cy < ty; ++cy)
    {
        for (int cx = max (0, 2 * x - D / 2); cx < tx; ++cx)
        {
            if(!isnan(src.ptr (cy)[cx]))
            {
                sum += src.ptr (cy)[cx] * gaussKernel[(ty - cy - 1) * 5 + (tx - cx - 1)];
                count += gaussKernel[(ty - cy - 1) * 5 + (tx - cx - 1)];
            }
        }
    }* /
    //dst.ptr (y)[x] = (float)(sum / (float)count);
    //data[y * 960 + x] = x / 960.0;
    //data[8] = 0.4;
    float1 test = make_float1(0.99);
    surf2Dwrite(test, tex, x*sizeof(float1), y);
}

//FIXME Remove
void testCuda(hipSurfaceObject_t surface)//(float* data)
{
    //dst.create (src.rows () / 2, src.cols () / 2);

    dim3 block (32, 8);
    dim3 grid (getGridDim (960, block.x), getGridDim (540, block.y));

    testKernel<<<grid, block>>>(surface);
    cudaCheckError();
}*/

void pyrDownGaussF(const DeviceArray2D<float>& src, DeviceArray2D<float> & dst)
{
    dst.create (src.rows () / 2, src.cols () / 2);

    dim3 block (32, 8);
    dim3 grid (getGridDim (dst.cols (), block.x), getGridDim (dst.rows (), block.y));

    const float gaussKernel[25] = {1, 4, 6, 4, 1,
                    4, 16, 24, 16, 4,
                    6, 24, 36, 24, 6,
                    4, 16, 24, 16, 4,
                    1, 4, 6, 4, 1};

    float * gauss_cuda;

    cudaSafeCall(hipMalloc((void**) &gauss_cuda, sizeof(float) * 25));
    cudaSafeCall(hipMemcpy(gauss_cuda, &gaussKernel[0], sizeof(float) * 25, hipMemcpyHostToDevice));

    pyrDownKernelGaussF<<<grid, block>>>(src, dst, gauss_cuda);
    cudaCheckError();

    hipFree(gauss_cuda);
}

__global__ void pyrDownKernelIntensityGauss(const PtrStepSz<unsigned char> src, PtrStepSz<unsigned char> dst, float * gaussKernel)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= dst.cols || y >= dst.rows)
        return;

    const int D = 5;

    int center = src.ptr (2 * y)[2 * x];

    int tx = min (2 * x - D / 2 + D, src.cols - 1);
    int ty = min (2 * y - D / 2 + D, src.rows - 1);
    int cy = max (0, 2 * y - D / 2);

    float sum = 0;
    int count = 0;

    for (; cy < ty; ++cy)
        for (int cx = max (0, 2 * x - D / 2); cx < tx; ++cx)
        {
            //This might not be right, but it stops incomplete model images from making up colors
            if(src.ptr (cy)[cx] > 0)
            {
                sum += src.ptr (cy)[cx] * gaussKernel[(ty - cy - 1) * 5 + (tx - cx - 1)];
                count += gaussKernel[(ty - cy - 1) * 5 + (tx - cx - 1)];
            }
        }
    dst.ptr (y)[x] = (sum / (float)count);
}

void pyrDownUcharGauss(const DeviceArray2D<unsigned char>& src, DeviceArray2D<unsigned char> & dst)
{
    dst.create (src.rows () / 2, src.cols () / 2);

    dim3 block (32, 8);
    dim3 grid (getGridDim (dst.cols (), block.x), getGridDim (dst.rows (), block.y));

    const float gaussKernel[25] = {1, 4, 6, 4, 1,
                    4, 16, 24, 16, 4,
                    6, 24, 36, 24, 6,
                    4, 16, 24, 16, 4,
                    1, 4, 6, 4, 1};

    float * gauss_cuda;

    hipMalloc((void**) &gauss_cuda, sizeof(float) * 25);
    hipMemcpy(gauss_cuda, &gaussKernel[0], sizeof(float) * 25, hipMemcpyHostToDevice);

    pyrDownKernelIntensityGauss<<<grid, block>>>(src, dst, gauss_cuda);
    cudaCheckError();

    hipFree(gauss_cuda);
}

/*void pyrDown2(const DeviceArray2D<unsigned char> & src, DeviceArray2D<unsigned char> & dst)
{
    dst.create (src.rows () / 2, src.cols () / 2);

    dim3 block (32, 8);
    dim3 grid (getGridDim (dst.cols (), block.x), getGridDim (dst.rows (), block.y));

    //pyrDownUcharGauss<<<grid, block>>>(src, dst);
    pyrDownUcharGauss()
    cudaCheckError();
}*/

__global__ void verticesToDepthKernel(const float * vmap_src, PtrStepSz<float> dst, float cutOff)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= dst.cols || y >= dst.rows)
        return;

    float z = vmap_src[y * dst.cols * 4 + (x * 4) + 2];

    dst.ptr(y)[x] = z > cutOff || z <= 0 ? __int_as_float(0x7fffffff)/*HIP_NAN_F*/ : z;
}

void verticesToDepth(DeviceArray<float>& vmap_src, DeviceArray2D<float> & dst, float cutOff)
{
    dim3 block (32, 8);
    dim3 grid (getGridDim (dst.cols (), block.x), getGridDim (dst.rows (), block.y));

    verticesToDepthKernel<<<grid, block>>>(vmap_src, dst, cutOff);
    cudaCheckError();
}

texture<uchar4, 2, hipReadModeElementType> inTex;

__global__ void bgr2IntensityKernel(PtrStepSz<unsigned char> dst)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= dst.cols || y >= dst.rows)
        return;

    uchar4 src = tex2D(inTex, x, y);

    int value = (float)src.x * 0.114f + (float)src.y * 0.299f + (float)src.z * 0.587f;

    printf("%d\n", value);

    dst.ptr (y)[x] = value;
}

void imageBGRToIntensity(hipArray * cuArr, DeviceArray2D<unsigned char> & dst)
{
    dim3 block (32, 8);
    dim3 grid (getGridDim (dst.cols (), block.x), getGridDim (dst.rows (), block.y));

    cudaSafeCall(hipBindTextureToArray(inTex, cuArr));

    bgr2IntensityKernel<<<grid, block>>>(dst);

    cudaCheckError();

    cudaSafeCall(hipUnbindTexture(inTex));
}

__global__ void bgr2IntensityKernelDMC3(int rows, int cols, float * rgb_src, PtrStepSz<unsigned char> rgb_dst)
{
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    if (x < cols && y < rows)
    {
        float3 vsrc, vdst = make_float3 (__int_as_float(0x7fffffff), __int_as_float(0x7fffffff), __int_as_float(0x7fffffff));
        vsrc.x = rgb_src[y * cols * 3 + (x * 3) + 0];
        vsrc.y = rgb_src[y * cols * 3 + (x * 3) + 1];
        vsrc.z = rgb_src[y * cols * 3 + (x * 3) + 2];
        int value = (float)vsrc.x * 0.114f + (float)vsrc.y * 0.299f + (float)vsrc.z * 0.587f;
        rgb_dst.ptr(y)[x] = value;

    }
}

__global__ void bgr2IntensityKernelDMC4(int rows, int cols, float * rgb_src, PtrStepSz<unsigned char> rgb_dst)
{
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    if (x < cols && y < rows)
    {
        float3 vsrc, vdst = make_float3 (__int_as_float(0x7fffffff), __int_as_float(0x7fffffff), __int_as_float(0x7fffffff));
        vsrc.x = rgb_src[y * cols * 4 + (x * 4) + 0];
        vsrc.y = rgb_src[y * cols * 4 + (x * 4) + 1];
        vsrc.z = rgb_src[y * cols * 4 + (x * 4) + 2];
        int value = (float)vsrc.x * 0.114f + (float)vsrc.y * 0.299f + (float)vsrc.z * 0.587f;
        rgb_dst.ptr(y)[x] = value;

    }
}

void imageBGRToIntensityDM(DeviceArray<float>& rgb_src, bool c3, DeviceArray2D<unsigned char>& rgb_dst)
{
    dim3 block (32, 8);
    dim3 grid (getGridDim (rgb_dst.cols(), block.x), getGridDim (rgb_dst.rows(), block.y));
    int rows = rgb_dst.rows(); // TO DO CHANGED changed / 3
    int cols = rgb_dst.cols();
    if (c3)
        bgr2IntensityKernelDMC3<<<grid, block>>>(rows, cols, rgb_src, rgb_dst);
    else
        bgr2IntensityKernelDMC4<<<grid, block>>>(rows, cols, rgb_src, rgb_dst);

    cudaCheckError();

}

__constant__ float gsobel_x3x3[9];
__constant__ float gsobel_y3x3[9];

__global__ void applyKernel(const PtrStepSz<unsigned char> src, PtrStep<short> dx, PtrStep<short> dy)
{

  int x = threadIdx.x + blockIdx.x * blockDim.x;
  int y = threadIdx.y + blockIdx.y * blockDim.y;

  if(x >= src.cols || y >= src.rows)
    return;

  float dxVal = 0;
  float dyVal = 0;

  int kernelIndex = 8;
  for(int j = max(y - 1, 0); j <= min(y + 1, src.rows - 1); j++)
  {
      for(int i = max(x - 1, 0); i <= min(x + 1, src.cols - 1); i++)
      {
          dxVal += (float)src.ptr(j)[i] * gsobel_x3x3[kernelIndex];
          dyVal += (float)src.ptr(j)[i] * gsobel_y3x3[kernelIndex];
          --kernelIndex;
      }
  }

  dx.ptr(y)[x] = dxVal;
  dy.ptr(y)[x] = dyVal;
}

void computeDerivativeImages(DeviceArray2D<unsigned char>& src, DeviceArray2D<short>& dx, DeviceArray2D<short>& dy)
{
    static bool once = false;

    if(!once)
    {
        float gsx3x3[9] = {0.52201,  0.00000, -0.52201,
                           0.79451, -0.00000, -0.79451,
                           0.52201,  0.00000, -0.52201};

        float gsy3x3[9] = {0.52201, 0.79451, 0.52201,
                           0.00000, 0.00000, 0.00000,
                           -0.52201, -0.79451, -0.52201};

        hipMemcpyToSymbol(HIP_SYMBOL(gsobel_x3x3), gsx3x3, sizeof(float) * 9);
        hipMemcpyToSymbol(HIP_SYMBOL(gsobel_y3x3), gsy3x3, sizeof(float) * 9);

        cudaSafeCall(hipGetLastError());
        cudaSafeCall(hipDeviceSynchronize());

        once = true;
    }

    dim3 block(32, 8);
    dim3 grid(getGridDim (src.cols (), block.x), getGridDim (src.rows (), block.y));

    applyKernel<<<grid, block>>>(src, dx, dy);

    cudaSafeCall(hipGetLastError());
    cudaSafeCall(hipDeviceSynchronize());
}


__global__ void projectPointsKernel(const PtrStepSz<float> depth,
                                    PtrStepSz<float3> cloud,
                                    const float invFx,
                                    const float invFy,
                                    const float cx,
                                    const float cy)
{
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;

    if (x >= depth.cols || y >= depth.rows)
        return;

    float z = depth.ptr(y)[x];

    cloud.ptr(y)[x].x = (float)((x - cx) * z * invFx);
    cloud.ptr(y)[x].y = (float)((y - cy) * z * invFy);
    cloud.ptr(y)[x].z = z;
}

void projectToPointCloud(const DeviceArray2D<float> & depth,
                         const DeviceArray2D<float3> & cloud,
                         CameraModel & intrinsics,
                         const int & level)
{
    dim3 block (32, 8);
    dim3 grid (getGridDim (depth.cols (), block.x), getGridDim (depth.rows (), block.y));

    CameraModel intrinsicsLevel = intrinsics(level);

    projectPointsKernel<<<grid, block>>>(depth, cloud, 1.0f / intrinsicsLevel.fx, 1.0f / intrinsicsLevel.fy, intrinsicsLevel.cx, intrinsicsLevel.cy);
    cudaCheckError();
    cudaSafeCall (hipDeviceSynchronize ());
}


__global__ void splatDepthPredictKernel(float cx, float cy, float fx, float fy, int rows, int cols, float maxDepth, float* tinv, float* model_buffer, /*PtrStepSz<float> color_dst*/float* color_dst, PtrStepSz<float> vmap_dst, PtrStepSz<float> nmap_dst, PtrStepSz<unsigned int> time_dst, int count)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int rows_mb, cols_mb;
    rows_mb = cols_mb = 3072;
    int vp_w, vp_h;
    vp_w = cols;
    vp_h = rows;
    
    if (i >= cols_mb * rows_mb)
        return;
    if (i >= count)
        return;
    if ((model_buffer[i] == 0) && (model_buffer[i + rows_mb*cols_mb] == 0) && (model_buffer[i + 2*rows_mb*cols_mb] == 0)) 
        return;

    // float4 vsrc = make_float4(__int_as_float(0x7fffffff), __int_as_float(0x7fffffff), __int_as_float(0x7fffffff), __int_as_float(0x7fffffff));
    // float4 nsrc = make_float4(__int_as_float(0x7fffffff), __int_as_float(0x7fffffff), __int_as_float(0x7fffffff), __int_as_float(0x7fffffff));

    float4 vsrc = make_float4(0, 0, 0, 0);
    float4 nsrc = make_float4(0, 0, 0, 0);

    //reading vertex and conf
    vsrc.x = model_buffer[i];
    vsrc.y = model_buffer[i + rows_mb*cols_mb];
    vsrc.z = model_buffer[i + 2*rows_mb*cols_mb];
    vsrc.w = model_buffer[i + 3*rows_mb*cols_mb];

    //reading normal and radius
    nsrc.x = model_buffer[i+8*rows_mb*cols_mb];
    nsrc.y = model_buffer[i+9*rows_mb*cols_mb];
    nsrc.z = model_buffer[i+10*rows_mb*cols_mb];
    nsrc.w = model_buffer[i+11*rows_mb*cols_mb];

    // printf("%f\n", model_buffer[i + rows_mb*cols_mb]);

    //reading color
    float c;
    c = model_buffer[i+4*rows_mb*cols_mb]; //x

    //reading time
    unsigned int t;
    t = (unsigned int)model_buffer[i+7*rows_mb*cols_mb];


    if (isnan (vsrc.x) || isnan(vsrc.y) || isnan(vsrc.z))
        return;
    if (isnan (nsrc.x) || isnan(nsrc.y) || isnan(nsrc.z))
        return;

    float3 v_ = make_float3(__int_as_float(0x7fffffff), __int_as_float(0x7fffffff), __int_as_float(0x7fffffff));
    float3 n_ = make_float3(__int_as_float(0x7fffffff), __int_as_float(0x7fffffff), __int_as_float(0x7fffffff));

    v_.x = tinv[0]*vsrc.x + tinv[1]*vsrc.y + tinv[2]*vsrc.z + tinv[3]*1;
    v_.y = tinv[4]*vsrc.x + tinv[5]*vsrc.y + tinv[6]*vsrc.z + tinv[7]*1;
    v_.z = tinv[8]*vsrc.x + tinv[9]*vsrc.y + tinv[10]*vsrc.z + tinv[11]*1;


    n_.x = tinv[0]*nsrc.x + tinv[1]*nsrc.y + tinv[2]*nsrc.z;
    n_.y = tinv[4]*nsrc.x + tinv[5]*nsrc.y + tinv[6]*nsrc.z;
    n_.z = tinv[8]*nsrc.x + tinv[9]*nsrc.y + tinv[10]*nsrc.z;
    n_ = normalized(n_);


    if (isnan (v_.x) || isnan(v_.y) || isnan(v_.z))
        return;
    if (isnan (n_.x) || isnan(n_.y) || isnan(n_.z))
        return;

    //to compute x,y cords (gl_fragcords)
    //TO DO need to normalize v_ 
    float3 fc;
    fc = projectPoint(v_, rows, cols, cx, cy, fx, fy, maxDepth);
    fc.x = fc.x * 0.5f + 0.5f; 
    fc.y = fc.y * 0.5f + 0.5f; 
    fc.x = fc.x * vp_w;
    fc.y = fc.y * vp_h;

    int x, y;
    x = (int)fc.x;
    y = (int)fc.y;

    float3 l = make_float3(__int_as_float(0x7fffffff), __int_as_float(0x7fffffff), __int_as_float(0x7fffffff));
    float3 cp = make_float3(__int_as_float(0x7fffffff), __int_as_float(0x7fffffff), __int_as_float(0x7fffffff));

    l.x = (fc.x - cx)/fx;
    l.y = (fc.y - cy)/fy;
    l.z = 1;
    l = normalized(l);

    float coeff;
    coeff = dot(v_, n_) / dot(l, n_);
    cp.x = l.x * coeff;
    cp.y = l.y * coeff;
    cp.z = l.z * coeff;

    float sqrRad = pow(nsrc.w, 2);
    float3 diff;
    diff.x = cp.x - vsrc.x;
    diff.y = cp.y - vsrc.y;
    diff.z = cp.z - vsrc.z;

    if (dot(diff,diff) > sqrRad)
    {
        return;
    }

    float3 dc; 
    dc = decodeColor(c);

    //writing color TO DO UPDATE disableing color decoding

    // color_dst[y*cols*4 + x*4 + 0] = dc.x;
    // color_dst[y*cols*4 + x*4 + 1] = dc.y;
    // color_dst[y*cols*4 + x*4 + 2] = dc.z;
    // color_dst[y*cols*4 + x*4 + 3] = 1;
    color_dst[y*cols*4 + x*4 + 0] = model_buffer[i+4*rows_mb*cols_mb];
    color_dst[y*cols*4 + x*4 + 1] = model_buffer[i+5*rows_mb*cols_mb];
    color_dst[y*cols*4 + x*4 + 2] = model_buffer[i+6*rows_mb*cols_mb];
    color_dst[y*cols*4 + x*4 + 3] = 1;

    //writing vertex and conf

    float3 sp = make_float3(0,0,0);
    sp.x = (fc.x - cx)*cp.z*(1/fx);
    sp.y = (fc.y - cy)*cp.z*(1/fy);
    sp.z = cp.z;

    if ((sp.z < 0) || (sp.z > maxDepth))
    {
        return;
    }
    // if ((sp.x < 0) || (sp.x > cols))
    // {
    //     return;
    // }
    // if ((sp.y < 0) || (sp.y > rows))
    // {
    //     return;
    // }

    vmap_dst.ptr(y)[x] = sp.x;
    vmap_dst.ptr(y + rows)[x] = sp.y;
    vmap_dst.ptr(y + rows * 2)[x] = sp.z;
    vmap_dst.ptr(y + rows * 3)[x] = vsrc.w;


    //writing normal and radius
    nmap_dst.ptr(y       )[x] = n_.x;
    nmap_dst.ptr(y + rows)[x] = n_.y;
    nmap_dst.ptr(y + 2 * rows)[x] = n_.z;
    nmap_dst.ptr(y + 3 * rows)[x] = nsrc.w;

    //writing time
    time_dst.ptr(y)[x] = t;

}

void splatDepthPredict(const CameraModel& intr, int rows, int cols, float maxDepth, float* pose_inv, DeviceArray<float>& model_buffer, int count,  /*DeviceArray2D<float>& color_dst*/DeviceArray<float>& color_dst, DeviceArray2D<float>& vmap_dst, DeviceArray2D<float>& nmap_dst, DeviceArray2D<unsigned int>& time_dst)
{
    int blocksize = 32*8;
    int numblocks = (count + blocksize - 1)/ blocksize;


    float* vertices = new float[rows*cols*4];
    memset(&vertices[0], 0, rows*cols*4);

    vmap_dst.create(rows*4, cols); // TO DO put it outside
    vmap_dst.upload(&vertices[0], sizeof(float)*cols, 4*rows, cols);

    // color_dst.create(rows*4, cols);
    // color_dst.upload(&vertices[0], sizeof(float)*cols, 4*rows, cols);

    color_dst.create(rows*4*cols);
    color_dst.upload(&vertices[0], rows*4*cols);

    nmap_dst.create(rows*4, cols);
    nmap_dst.upload(&vertices[0], sizeof(float)*cols, 4*rows, cols);
    
    time_dst.create(rows,cols);
    time_dst.upload(&vertices[0], sizeof(float)*cols, rows, cols);

    delete[] vertices;

    float fx = intr.fx, cx = intr.cx;
    float fy = intr.fy, cy = intr.cy;

    float * tinv;
    cudaSafeCall(hipMalloc((void**) &tinv, sizeof(float) * 16));
    cudaSafeCall(hipMemcpy(tinv, pose_inv, sizeof(float) * 16, hipMemcpyHostToDevice));

    // std::cout<<"cx = "<<cx<<": cy = "<<cy<<": fx = "<<fx<<": fy = "<<fy<<": rows = "<<rows<<": cols = "<<cols;
    splatDepthPredictKernel<<<numblocks, blocksize>>>(cx, cy, fx, fy, rows, cols, maxDepth, tinv, model_buffer, color_dst, vmap_dst, nmap_dst, time_dst, count);
    cudaCheckError();

}

__global__ void predictIndiciesKernel(float cx, float cy, float fx, float fy,  int rows, int cols, float maxDepth, float* tinv, float* model_buffer, int time, int timeDelta, PtrStepSz<float> vmap_pi, PtrStepSz<float> ct_pi, PtrStepSz<float> nmap_pi, PtrStepSz<unsigned int> index_pi)
{

    int i = threadIdx.x + blockIdx.x * blockDim.x;

    int rows_mb, cols_mb;
    rows_mb = cols_mb = 3072;
    // int i = y* rows + x;
    float xu = 0;
    float yv = 0;

    if (i >= rows_mb*cols_mb)
        return;

    int vz = model_buffer[i + 2*rows_mb*cols_mb];
    int cw = model_buffer[i+7*rows_mb*cols_mb];
    int vertexId;
    float3 vsrc = make_float3(__int_as_float(0x7fffffff), __int_as_float(0x7fffffff), __int_as_float(0x7fffffff));

    if ((vz < 0 ) || (vz > maxDepth) || (time - cw > timeDelta))
    {
        vsrc.x = -10;
        vsrc.y = -10;
        vertexId = 0;
    }
    else
    {
        float3 v_ = make_float3(__int_as_float(0x7fffffff), __int_as_float(0x7fffffff), __int_as_float(0x7fffffff));
        float3 nsrc = make_float3(__int_as_float(0x7fffffff), __int_as_float(0x7fffffff), __int_as_float(0x7fffffff));
        float3 n_ = make_float3(__int_as_float(0x7fffffff), __int_as_float(0x7fffffff), __int_as_float(0x7fffffff));

        vsrc.x = model_buffer[i];
        vsrc.y = model_buffer[i + rows_mb*cols_mb];
        vsrc.z = model_buffer[i + 2*rows_mb*cols_mb];

        nsrc.x = model_buffer[i+8*rows_mb*cols_mb];
        nsrc.y = model_buffer[i+9*rows_mb*cols_mb];
        nsrc.z = model_buffer[i+10*rows_mb*cols_mb];

        v_.x = tinv[0]*vsrc.x + tinv[1]*vsrc.y + tinv[2]*vsrc.z + tinv[3]*1;
        v_.y = tinv[4]*vsrc.x + tinv[5]*vsrc.y + tinv[6]*vsrc.z + tinv[7]*1;
        v_.z = tinv[8]*vsrc.x + tinv[9]*vsrc.y + tinv[10]*vsrc.z + tinv[11]*1;

        // xu = ((((fx* v_.x) / v_.z) + cx) - (cols * 0.5)) / (cols * 0.5);
        // yv = ((((fy * v_.y) / v_.z) + cy) - (rows * 0.5)) / (rows * 0.5);
        // vertexId = gl_VertexID;
        vertexId = i;

        n_.x = tinv[0]*nsrc.x + tinv[1]*nsrc.y + tinv[2]*nsrc.z;
        n_.y = tinv[4]*nsrc.x + tinv[5]*nsrc.y + tinv[6]*nsrc.z;
        n_.z = tinv[8]*nsrc.x + tinv[9]*nsrc.y + tinv[10]*nsrc.z;
        n_ = normalized(n_);


        float3 fc;
        fc = projectPoint(v_, rows, cols, cx, cy, fx, fy, maxDepth);
        fc.x = fc.x * 0.5f + 0.5f; 
        fc.y = fc.y * 0.5f + 0.5f; 
        fc.x = fc.x * cols;
        fc.y = fc.y * rows;
        int x = fc.x, y = fc.y;

        // printf("x = %d y = %d\n", x, y);
        vmap_pi.ptr(y)[x] = v_.x;
        vmap_pi.ptr(y + rows)[x] = v_.y;
        vmap_pi.ptr(y + rows * 2)[x] = v_.z;
        vmap_pi.ptr(y + rows * 3)[x] = model_buffer[i + 3*rows_mb*cols_mb];

        ct_pi.ptr(y)[x] = model_buffer[i+4*rows_mb*cols_mb];
        ct_pi.ptr(y + rows)[x] = model_buffer[i+5*rows_mb*cols_mb];
        ct_pi.ptr(y + rows * 2)[x] = model_buffer[i+6*rows_mb*cols_mb];
        ct_pi.ptr(y + rows * 3)[x] = model_buffer[i+7*rows_mb*cols_mb];

        nmap_pi.ptr(y)[x] = n_.x;
        nmap_pi.ptr(y + rows)[x] = n_.y;
        nmap_pi.ptr(y + rows * 2)[x] = n_.z;
        nmap_pi.ptr(y + rows * 3)[x] = model_buffer[i + 11*rows_mb*cols_mb];

        index_pi.ptr(y)[x] = i;
    }
}

void predictIndicies(const CameraModel& intr, int rows, int cols, float maxDepth, float* pose_inv, DeviceArray<float>& model_buffer, int time, DeviceArray2D<float>& vmap_pi, DeviceArray2D<float>& ct_pi, DeviceArray2D<float>& nmap_pi, DeviceArray2D<unsigned int>& index_pi, int count)
{
    int blocksize = 32*8;
    int numblocks = (count + blocksize - 1)/ blocksize;
    int timeDelta = 200;

    float* vertices = new float[rows*cols*4];
    memset(&vertices[0], 0, rows*cols*4);

    vmap_pi.create(rows*4, cols); 
    vmap_pi.upload(&vertices[0], sizeof(float)*cols, 4*rows, cols);

    ct_pi.create(rows*4, cols);
    ct_pi.upload(&vertices[0], sizeof(float)*cols, 4*rows, cols);

    nmap_pi.create(rows*4, cols);
    nmap_pi.upload(&vertices[0], sizeof(float)*cols, 4*rows, cols);
    
    index_pi.create(rows,cols);
    index_pi.upload(&vertices[0], sizeof(float)*cols, rows, cols);

    delete[] vertices;

    float fx = intr.fx, cx = intr.cx;
    float fy = intr.fy, cy = intr.cy;

    float * tinv;
    cudaSafeCall(hipMalloc((void**) &tinv, sizeof(float) * 16));
    cudaSafeCall(hipMemcpy(tinv, pose_inv, sizeof(float) * 16, hipMemcpyHostToDevice));

    predictIndiciesKernel<<<numblocks, blocksize>>>(cx, cy, fx, fy, rows, cols, maxDepth, tinv, model_buffer, time, timeDelta, vmap_pi, ct_pi, nmap_pi, index_pi);

}

__device__ bool checkNeighbours(const PtrStepSz<float> depth, int u ,int v)
{

    float z = depth.ptr(v)[u + 1];
    if(z == 0)
        return false;

    z = depth.ptr(v)[u - 1];
    if(z == 0)
        return false;

    z = depth.ptr(v - 1)[u];
    if(z == 0)
        return false;

    z = depth.ptr(v + 1)[u];
    if(z == 0)
        return false;

    return true;
}

__device__ float angleBetween(float3 a, float3 b)
{
    return acos(dot(a, b) / (sqrt(pow(a.x,2)+pow(a.y,2)+pow(a.z,2)) * sqrt(pow(b.x,2)+pow(b.y,2)+pow(b.z,2))));
}

__global__ void fuseKernel(const PtrStepSz<float> depth, const float* rgb, const PtrStepSz<float> depthf, float cx, float cy, float fx, float fy, int rows, int cols, float maxDepth, float* pose, float* model_buffer, int time, PtrStepSz<float> vmap_pi, PtrStepSz<float> ct_pi, PtrStepSz<float> nmap_pi, PtrStepSz<unsigned int> index_pi, int* count, float weighting)
{

    int u = threadIdx.x + blockIdx.x * blockDim.x;
    int v = threadIdx.y + blockIdx.y * blockDim.y;

    int i = v*cols + u;

    int rows_mb, cols_mb;
    rows_mb = cols_mb = 3072;

    if(u < depth.cols && v < depth.rows)
    {
        float z = depth.ptr(v)[u] /*/ 1000.f*/; // load and convert: mm -> meters

        if(z != 0 && z < maxDepth /*&& m == maskID*/) //FIXME
        {
            float3 vsrc_new = make_float3(z * (u - cx) / fx,  z * (v - cy) / fy,  z);
            float3 vnew_ = make_float3(0,0,0);
            vnew_.x = pose[0]*vsrc_new.x + pose[1]*vsrc_new.y + pose[2]*vsrc_new.z + pose[3]*1;
            vnew_.y = pose[4]*vsrc_new.x + pose[5]*vsrc_new.y + pose[6]*vsrc_new.z + pose[7]*1;
            vnew_.z = pose[8]*vsrc_new.x + pose[9]*vsrc_new.y + pose[10]*vsrc_new.z + pose[11]*1;

            float zf = depthf.ptr(v)[u];
            float3 vsrc_new_f = make_float3(z * (u - cx) / fx,  z * (v - cy) / fy,  zf); //depth filtering look up
            
            float3 cnew = make_float3(0,0,0);
            float cvw = 0;
            float ec_new ;
            cnew.x = rgb[i];
            cnew.y = rgb[i + rows*cols];
            cnew.z = rgb[i + 2*rows*cols];
            ec_new = encodeColor(cnew);
            
            float3 nsrc_new = make_float3(0,0,0);
            float4 nnew_ = make_float4(0,0,0,0);
            float rnew;
            nsrc_new = getNormal(depth, vsrc_new_f, cx, cy ,fx ,fy, u, v, rows, cols); // TO change vsrc_new  to vsrc_new_f
            nnew_.x = pose[0]*nsrc_new.x + pose[1]*nsrc_new.y + pose[2]*nsrc_new.z;
            nnew_.y = pose[4]*nsrc_new.x + pose[5]*nsrc_new.y + pose[6]*nsrc_new.z;
            nnew_.z = pose[8]*nsrc_new.x + pose[9]*nsrc_new.y + pose[10]*nsrc_new.z;
            nnew_.w =  getRadius(fx, fy, vsrc_new_f.z, nsrc_new.z); // TO DO change vsrc_new.z to vsrc_new_f.z

            // //Confidence
            // vPosition.w = confidence(x, y, weighting);

            float confnew = confidence(cx, cy, u, v, weighting);
            unsigned int best = 0U;
            int operation = 0;
            float4 vPosition0, vNormRad0, vColor0, vPosition, vNormRad, vColor;
            float c_k, a;
            float3 v_k, v_g;

            if((int(u) % 2 == int(time) % 2) && (int(v) % 2 == int(time) % 2) && checkNeighbours(depth, u, v) && vsrc_new.z > 0 && vsrc_new.z <= maxDepth)
            {
                float bestDist = 1000;

                float xl = (u - cx) * fx;
                float yl = (v - cy) * fy;

                float lambda = sqrt(xl * xl + yl * yl + 1);
                float3 ray = make_float3(xl, yl, 1);


                for (int ui = u - 2; ui < u + 2; ui++)
                {
                    for (int vj = v - 2; vj < v + 2; vj++)
                    {
                        if ((ui < 0) || (ui >=cols))
                            continue;
                        if ((vj < 0) || (vj >=rows))
                            continue;
                        unsigned int current = index_pi.ptr(vj)[ui];
                        if(current > 0U)
                        {
                            float4 vertConf = make_float4(0,0,0,0);
                            vertConf.x = vmap_pi.ptr(vj)[ui];
                            vertConf.y = vmap_pi.ptr(vj + rows)[ui];
                            vertConf.z = vmap_pi.ptr(vj + rows * 2)[ui];
                            vertConf.w = vmap_pi.ptr(vj + rows * 3)[ui];


                            vPosition = make_float4(vmap_pi.ptr(v)[u], vmap_pi.ptr(v + rows)[u], vmap_pi.ptr(v + rows * 2)[u], vmap_pi.ptr(v + rows * 3)[u]);
                            vNormRad =  make_float4(nmap_pi.ptr(v)[u], nmap_pi.ptr(v + rows)[u], nmap_pi.ptr(v + rows * 2)[u], nmap_pi.ptr(v + rows * 3)[u]);
                            vColor = make_float4(ct_pi.ptr(v)[u], ct_pi.ptr(v + rows)[u], ct_pi.ptr(v + rows * 2)[u], ct_pi.ptr(v + rows * 3)[u]);
                             
                            float zdiff = vertConf.z - vsrc_new.z;

                            if (abs(zdiff * lambda) < 0.05)
                            {
                                float3 ray_v_cross = make_float3(0,0,0);
                                ray_v_cross = cross(ray, make_float3(vertConf.x,vertConf.y,vertConf.z));
                                float dist = sqrt(pow(ray_v_cross.x,2) + pow(ray_v_cross.y,2) + pow(ray_v_cross.z,2)) / lambda;
                                float4 normRad = make_float4(0,0,0,0);
                                normRad.x = nmap_pi.ptr(vj)[ui];
                                normRad.y = nmap_pi.ptr(vj + rows)[ui];
                                normRad.z = nmap_pi.ptr(vj + rows * 2)[ui];
                                normRad.w = nmap_pi.ptr(vj + rows * 3)[ui];
                                
                                float abw = angleBetween(make_float3(normRad.x, normRad.y, normRad.z), make_float3(nnew_.x, nnew_.y, nnew_.z));
                                
                                if(dist < bestDist && (abs(normRad.z) < 0.75f || abw < 0.5f))
                                {

                                    c_k = vPosition.w;
                                    v_k = make_float3(vPosition.x, vPosition.y, vPosition.z);

                                    a = confnew;
                                    v_g = vnew_;

                                    if(nnew_.w < (1.0 + 0.5) * vNormRad.w)
                                    {
                                        operation = 1;
                                        bestDist = dist;
                                        best = current;
                                    }
                                }
                            }
                        }
                    }
                }
                if (operation == 1)
                {
                    vPosition0 = make_float4((c_k * v_k.x + a * v_g.x) / (c_k + a),(c_k * v_k.y + a * v_g.y) / (c_k + a),(c_k * v_k.z + a * v_g.z) / (c_k + a),
                                      c_k + a); // Add up confidence, weighted position
                    float3 oldCol = decodeColor(vColor.x);
                    float3 newCol = decodeColor(ec_new);
                    float3 avgColor = make_float3((c_k * oldCol.x+ a * newCol.x)/ (c_k + a), (c_k * oldCol.y+ a * newCol.y)/ (c_k + a), (c_k * oldCol.z+ a * newCol.z)/ (c_k + a));
                    vColor0 = make_float4(encodeColor(avgColor), vColor.y, vColor.z, time);
                    vNormRad0 = make_float4((c_k * vNormRad.x+ a * nnew_.x)/ (c_k + a), (c_k * vNormRad.y+ a * nnew_.y)/ (c_k + a), (c_k * vNormRad.z+ a * nnew_.z)/ (c_k + a), (c_k * vNormRad.w+ a * nnew_.w)/ (c_k + a));
                    float3 normnrad = normalized(make_float3(vNormRad0.x,vNormRad0.y,vNormRad0.z));
                    vNormRad0.x = normnrad.x;
                    vNormRad0.y = normnrad.y;
                    vNormRad0.z = normnrad.z;

                    //writing vertex and confidence
                    model_buffer[best] = vPosition0.x;
                    model_buffer[best + rows_mb*cols_mb] = vPosition0.y;
                    model_buffer[best + 2*rows_mb*cols_mb] = vPosition0.z;
                    model_buffer[best + 3*rows_mb*cols_mb] = vPosition0.w;

                    //writing color and time
                    model_buffer[best + 4*rows_mb*cols_mb] = vColor0.x; //x
                    model_buffer[best + 5*rows_mb*cols_mb] = vColor0.y;//y
                    model_buffer[best + 6*rows_mb*cols_mb] = vColor0.z;//z
                    model_buffer[best + 7*rows_mb*cols_mb] = vColor0.w;//w time

                    //writing normals
                    model_buffer[best + 8*rows_mb*cols_mb] = vNormRad0.x;
                    model_buffer[best + 9*rows_mb*cols_mb] = vNormRad0.y;
                    model_buffer[best + 10*rows_mb*cols_mb] = vNormRad0.z;
                    model_buffer[best + 11*rows_mb*cols_mb] = vNormRad0.w;

                }
                else
                {
                    // vPosition0 = vPosition;
                    // vColor0 = vColor;
                    // vNormRad0 = vNormRad;
                    // vPosition0.w = c_k + a;
                    // vColor0.w = time;

                    // writing vertex and confidence
                    // model_buffer[*count] = vPosition0.x;
                    // model_buffer[*count + rows_mb*cols_mb] = vPosition0.y;
                    // model_buffer[*count + 2*rows_mb*cols_mb] = vPosition0.z;
                    // model_buffer[*count + 3*rows_mb*cols_mb] = vPosition0.w;

                    // //writing color and time
                    // model_buffer[*count + 4*rows_mb*cols_mb] = vColor0.x; //x
                    // model_buffer[*count + 5*rows_mb*cols_mb] = vColor0.y;//y
                    // model_buffer[*count + 6*rows_mb*cols_mb] = vColor0.z;//z
                    // model_buffer[*count + 7*rows_mb*cols_mb] = vColor0.w;//w time

                    // // writing normals
                    // model_buffer[*count + 8*rows_mb*cols_mb] = vNormRad0.x;
                    // model_buffer[*count + 9*rows_mb*cols_mb] = vNormRad0.y;
                    // model_buffer[*count + 10*rows_mb*cols_mb] = vNormRad0.z;
                    // model_buffer[*count + 11*rows_mb*cols_mb] = vNormRad0.w;
                    // atomicAdd(count, 1);

                }
            }
        }
    }
}

void fuse(DeviceArray2D<float>& depth,  DeviceArray<float>& rgb, DeviceArray2D<float>& depthf, const CameraModel& intr, int rows, int cols, float maxDepth, float* pose, DeviceArray<float>& model_buffer, int * h_count, int time, DeviceArray2D<float>& vmap_pi, DeviceArray2D<float>& ct_pi, DeviceArray2D<float>& nmap_pi, DeviceArray2D<unsigned int>& index_pi, float weighting)
{
    dim3 block (32, 8);
    dim3 grid (1, 1, 1);
    grid.x = getGridDim (depth.cols (), block.x);
    grid.y = getGridDim (depth.rows (), block.y);

    float fx = intr.fx, cx = intr.cx;
    float fy = intr.fy, cy = intr.cy;

    int *d_count;
    hipMalloc((void**)&d_count, sizeof(int));
    hipMemcpy(d_count, h_count, sizeof(int), hipMemcpyHostToDevice);

    float * t;
    cudaSafeCall(hipMalloc((void**) &t, sizeof(float) * 16));
    cudaSafeCall(hipMemcpy(t, pose, sizeof(float) * 16, hipMemcpyHostToDevice));

    fuseKernel<<<grid, block>>>(depth, rgb, depthf, fx, fy, cx, cy, rows, cols, maxDepth, t, model_buffer, time, vmap_pi, ct_pi, nmap_pi, index_pi, d_count, weighting);
    cudaSafeCall(hipGetLastError());
    hipMemcpy(h_count, d_count, sizeof(int), hipMemcpyDeviceToHost);

}


// __global__ void DATestKernel(float* da_src, float* da_dst)
// {
    
// }