#include "hip/hip_runtime.h"
/*
 * This file is part of ElasticFusion.
 *
 * Copyright (C) 2015 Imperial College London
 * 
 * The use of the code within this file and all code within files that 
 * make up the software that is ElasticFusion is permitted for 
 * non-commercial purposes only.  The full terms and conditions that 
 * apply to the code within this file are detailed within the LICENSE.txt 
 * file and at <http://www.imperial.ac.uk/dyson-robotics-lab/downloads/elastic-fusion/elastic-fusion-license/> 
 * unless explicitly stated.  By downloading this file you agree to 
 * comply with these terms.
 *
 * If you wish to use any of this code for commercial purposes then 
 * please email researchcontracts.engineering@imperial.ac.uk.
 *
 * Software License Agreement (BSD License)
 *
 *  Copyright (c) 2011, Willow Garage, Inc.
 *  All rights reserved.
 *
 *  Redistribution and use in source and binary forms, with or without
 *  modification, are permitted provided that the following conditions
 *  are met:
 *
 *   * Redistributions of source code must retain the above copyright
 *     notice, this list of conditions and the following disclaimer.
 *   * Redistributions in binary form must reproduce the above
 *     copyright notice, this list of conditions and the following
 *     disclaimer in the documentation and/or other materials provided
 *     with the distribution.
 *   * Neither the name of Willow Garage, Inc. nor the names of its
 *     contributors may be used to endorse or promote products derived
 *     from this software without specific prior written permission.
 *
 *  THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS
 *  "AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT
 *  LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS
 *  FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE
 *  COPYRIGHT OWNER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT,
 *  INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING,
 *  BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 *  LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 *  CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT
 *  LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN
 *  ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
 *  POSSIBILITY OF SUCH DAMAGE.
 *
 *  Author: Anatoly Baskeheev, Itseez Ltd, (myname.mysurname@mycompany.com)
 */

#include "cudafuncs.cuh"
#include "convenience.cuh"
#include "operators.cuh"

__global__ void pyrDownGaussKernel (const PtrStepSz<float> src, PtrStepSz<float> dst, float sigma_color)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= dst.cols || y >= dst.rows)
        return;

    const int D = 5;

    int center = src.ptr (2 * y)[2 * x];

    int x_mi = max(0, 2*x - D/2) - 2*x;
    int y_mi = max(0, 2*y - D/2) - 2*y;

    int x_ma = min(src.cols, 2*x -D/2+D) - 2*x;
    int y_ma = min(src.rows, 2*y -D/2+D) - 2*y;

    float sum = 0;
    float wall = 0;

    float weights[] = {0.375f, 0.25f, 0.0625f} ;

    for(int yi = y_mi; yi < y_ma; ++yi)
        for(int xi = x_mi; xi < x_ma; ++xi)
        {
            int val = src.ptr (2*y + yi)[2*x + xi];

            if (abs (val - center) < 3 * sigma_color)
            {
                sum += val * weights[abs(xi)] * weights[abs(yi)];
                wall += weights[abs(xi)] * weights[abs(yi)];
            }
        }


    dst.ptr (y)[x] = static_cast<int>(sum / wall);
}

void pyrDown(const DeviceArray2D<unsigned short> & src, DeviceArray2D<unsigned short> & dst)
{
    dst.create (src.rows () / 2, src.cols () / 2);

    dim3 block (32, 8);
    dim3 grid (getGridDim (dst.cols (), block.x), getGridDim (dst.rows (), block.y));

    const float sigma_color = 30;

    pyrDownGaussKernel<<<grid, block>>>(src, dst, sigma_color);
    cudaCheckError();
}

__device__ float3 projectPoint(float3 p, int rows, int cols, float cx, float cy, float fx, float fy, float maxDepth)
{
    float3 pt = make_float3(
                ((((fx * p.x) / p.z) + cx) - (cols * 0.5)) / (cols * 0.5),
                ((((fy * p.y) / p.z) + cy) - (rows * 0.5)) / (rows * 0.5),
                p.z / maxDepth
                );
    return pt;
}

// Generate a vertex map 'vmap' based on the depth map 'depth' and camera parameters
__global__ void computeVmapKernel(const PtrStepSz<float> depth, PtrStep<float> vmap, float fx_inv, float fy_inv, float cx, float cy, float depthCutoff)
{
    int u = threadIdx.x + blockIdx.x * blockDim.x;
    int v = threadIdx.y + blockIdx.y * blockDim.y;

    if(u < depth.cols && v < depth.rows)
    {
        float z = depth.ptr(v)[u] /*/ 1000.f*/; // load and convert: mm -> meters

        if(z != 0 && z < depthCutoff /*&& m == maskID*/) //FIXME
        {
            float vx = z * (u - cx) * fx_inv;
            float vy = z * (v - cy) * fy_inv;
            float vz = z;

            vmap.ptr (v                 )[u] = vx;
            vmap.ptr (v + depth.rows    )[u] = vy;
            vmap.ptr (v + depth.rows * 2)[u] = vz;

            float3 p = make_float3(vx, vy, vz);
            p = projectPoint(p, depth.rows, depth.cols, cx, cy, 1/fx_inv, 1/fy_inv, depthCutoff);
            // printf(" p.x = %f\n", p.x);

        }
        else
        {
            vmap.ptr (v)[u] = __int_as_float(0x7fffffff); /*HIP_NAN_F*/
        }
    }
}


void createVMap(const CameraModel& intr, const DeviceArray2D<float> & depth, DeviceArray2D<float> & vmap, const float depthCutoff)
{
    vmap.create (depth.rows () * 3, depth.cols ());

    dim3 block (32, 8);
    dim3 grid (1, 1, 1);
    grid.x = getGridDim (depth.cols (), block.x);
    grid.y = getGridDim (depth.rows (), block.y);

    float fx = intr.fx, cx = intr.cx;
    float fy = intr.fy, cy = intr.cy;

    computeVmapKernel<<<grid, block>>>(depth, vmap, 1.f / fx, 1.f / fy, cx, cy, depthCutoff);
    cudaSafeCall(hipGetLastError());
}

__device__ float getRadius(float fx, float fy, float depth, float norm_z)
{
    float meanFocal = ((1.0 / abs(fx)) + (1.0 / abs(fy))) / 2.0;
    
    const float sqrt2 = 1.41421356237f;
    
    float radius = (depth / meanFocal) * sqrt2;

    float radius_n = radius;

    radius_n = radius_n / abs(norm_z);

    radius_n = min(2.0f * radius, radius_n);

    return radius_n;
}

__device__ float3 getNormal(const PtrStepSz<float> depth, float depthCutoff, float3 vpos, float cx, float cy, float fx, float fy, int u, int v, int rows, int cols)
{
    // if (!isnan (v00.x) && !isnan (v01.x) && !isnan (v10.x)) TO DO check for nan
    float z1 = depth.ptr(v)[u + 1];
    float z2 = depth.ptr(v)[u - 1];
    float z3 = depth.ptr(v + 1)[u];
    float z4 = depth.ptr(v - 1)[u];
    

    if(z1 != 0 && z1 < depthCutoff && z2 != 0 && z2 < depthCutoff && z3 != 0 && z3 < depthCutoff && z4 != 0 && z4 < depthCutoff)
    {

        float3 v1 = make_float3(z1 * (u + 1 - cx) / fx,  z1 * (v - cy) / fy,  z1); 
        float3 v2 = make_float3(z2 * (u - 1 - cx) / fx,  z2 * (v - cy) / fy,  z2); 
        float3 v3 = make_float3(z3 * (u - cx) / fx,  z3 * (v + 1 - cy) / fy,  z3); 
        float3 v4 = make_float3(z4 * (u - cx) / fx,  z4 * (v - 1- cy) / fy,  z4); 
        
        if (!isnan (v1.x) && !isnan (v2.x) && !isnan (v3.x))
        {
            float3 del_x = make_float3(((vpos.x + v2.x)/2 - (vpos.x + v1.x)/2), ((vpos.y + v2.y)/2 - (vpos.y + v1.y)/2), ((vpos.z + v2.z)/2 - (vpos.z + v1.z)/2));
            float3 del_y = make_float3(((vpos.x + v4.x)/2 - (vpos.x + v3.x)/2), ((vpos.y + v4.y)/2 - (vpos.y + v3.y)/2), ((vpos.z + v4.z)/2 - (vpos.z + v3.z)/2));
            float3 ret = normalized(cross(del_x, del_y));    
            return ret;
        }
    }
    else
        return make_float3(0,0,0); // TO DO not sure if putting 0s is right
}


__device__ float encodeColor(float3 c)
{
    int rgb = 0;
    rgb = int(round(c.x * 255.0f));
    rgb = (rgb << 8) + int(round(c.y * 255.0f));
    rgb = (rgb << 8) + int(round(c.z * 255.0f));
    return  (float)rgb;
}

__device__ float3 decodeColor(float c)
{
    float3 col;
    col.x = float(int(c) >> 16 & 0xFF) / 255.0f;
    col.y = float(int(c) >> 8 & 0xFF) / 255.0f;
    col.z = float(int(c) & 0xFF) / 255.0f;
    return col;
}


__device__ float confidence(float cx, float cy, float x, float y, float weighting)
{
    const float maxRadDist = 400; //sqrt((width * 0.5)^2 + (height * 0.5)^2)
    const float twoSigmaSquared = 0.72; //2*(0.6^2) from paper

    float3 pixelPosCentered = make_float3(x-cx, y-cy, 0);
    // vec2 pixelPosCentered = vec2(x, y) - cam.xy;
    float radialDist = sqrt(dot(pixelPosCentered, pixelPosCentered)) / maxRadDist;
    return exp((-(radialDist * radialDist) / twoSigmaSquared)) * weighting;
}


__global__ void initModelBufferKernel(float cx, float cy, float fx, float fy, int rows, int cols, float max_depth, float* model_buffer, const mat33 Rmat, const float3 tvec, int* count, const PtrStepSz<float> vmap, const PtrStepSz<float> nmap, const float* rgb)
{
    int u = threadIdx.x + blockIdx.x * blockDim.x;
    int v = threadIdx.y + blockIdx.y * blockDim.y;

    float vz = vmap.ptr(v + rows*2)[u];

    if ((vz < 0) || (vz > max_depth))
    {
        return;
    }

    if (u > 0 && u < cols && v > 0 && v < rows)
    {
        // replace this, hardcoding temporarily
        int rows_mb, cols_mb;
        rows_mb = cols_mb = 3072;
        int i = cols*v + u;

        float3 vert, vert_dst, norm, norm_dst;
        vert = make_float3(vmap.ptr(v)[u], vmap.ptr(v + rows)[u], vz);
        norm = make_float3(nmap.ptr(v)[u], nmap.ptr(v + rows)[u], nmap.ptr(v + rows*2)[u]);
        vert_dst = make_float3(0,0,0);
        norm_dst = make_float3(0,0,0);

        if (isnan(vert.x) || isnan(vert.y) || isnan(vert.z) || isnan(norm.x) || isnan(norm.y) || isnan(norm.z))
            return;


        vert_dst = Rmat * vert + tvec;
        norm_dst = Rmat * norm;
        // vert.x = pose[0]*vmap.ptr(v)[u] + pose[1]*vmap.ptr(v + rows)[u] + pose[2]*vz + pose[3]*1;
        // vert.y = pose[4]*vmap.ptr(v)[u] + pose[5]*vmap.ptr(v + rows)[u] + pose[6]*vz + pose[7]*1;
        // vert.z = pose[8]*vmap.ptr(v)[u] + pose[9]*vmap.ptr(v + rows)[u] + pose[10]*vz + pose[11]*1;
    
        // norm.x = pose[0]*nmap.ptr(v)[u] + pose[1]*nmap.ptr(v + rows)[u] + pose[2]*nmap.ptr(v + rows*2)[u];
        // norm.y = pose[4]*nmap.ptr(v)[u] + pose[5]*nmap.ptr(v + rows)[u] + pose[6]*nmap.ptr(v + rows*2)[u];
        // norm.z = pose[8]*nmap.ptr(v)[u] + pose[9]*nmap.ptr(v + rows)[u] + pose[10]*nmap.ptr(v + rows*2)[u];

        //writing vertex and confidence
        model_buffer[i] = vert_dst.x;
        model_buffer[i+ rows_mb*cols_mb] = vert_dst.y;
        model_buffer[i+2*rows_mb*cols_mb] = vert_dst.z;
        model_buffer[i+3*rows_mb*cols_mb] = confidence(cx, cy, u, v, 1);

        // color encoding
        float3 c;
        float ec ;
        c.x = rgb[v*cols*3 + u*3 + 0];
        c.y = rgb[v*cols*3 + u*3 + 1];
        c.z = rgb[v*cols*3 + u*3 + 2];

        c.x = c.x/255;
        c.y = c.y/255;
        c.z = c.z/255;

        ec = encodeColor(c);
        // //writing color and time
        model_buffer[i+4*rows_mb*cols_mb] = ec; //x
        model_buffer[i+5*rows_mb*cols_mb] = 0;//y
        model_buffer[i+6*rows_mb*cols_mb] = 1;//z
        model_buffer[i+7*rows_mb*cols_mb] = 1;//w time

        //writing normals
        model_buffer[i+8*rows_mb*cols_mb] = norm_dst.x;
        model_buffer[i+9*rows_mb*cols_mb] = norm_dst.y;
        model_buffer[i+10*rows_mb*cols_mb] = norm_dst.z;
        model_buffer[i+11*rows_mb*cols_mb] = getRadius(fx, fy, vmap.ptr(v + rows*2)[u], nmap.ptr(v + rows*2)[u]);
        
        atomicAdd(count, 1);
        // printf("initModelBuffer :vx = %f vy = %f vz = %f cx = %f cy = %f cz = %f nx = %f ny = %f nz = %f \n",model_buffer[i],model_buffer[i+ rows_mb*cols_mb], model_buffer[i+ 2*rows_mb*cols_mb], model_buffer[i+ 4*rows_mb*cols_mb], model_buffer[i+ 5*rows_mb*cols_mb], model_buffer[i+ 6*rows_mb*cols_mb], model_buffer[i+ 8*rows_mb*cols_mb], model_buffer[i+ 9*rows_mb*cols_mb], model_buffer[i+ 10*rows_mb*cols_mb]);
    }
}

void initModelBuffer(const CameraModel& intr, const float depthCutOff, DeviceArray<float> & model_buffer, const mat33& Rmat, const float3& tvec, int* h_count, const DeviceArray2D<float> & vmap, const DeviceArray2D<float> & nmap, const DeviceArray<float> & rgb )
{
    int *d_count;
    hipMalloc((void**)&d_count, sizeof(int));
    hipMemcpy(d_count, h_count, sizeof(int), hipMemcpyHostToDevice);

    int cols, rows;
    rows = vmap.rows()/3;
    cols = vmap.cols();
    dim3 block (32, 8);
    dim3 grid (1, 1, 1);
    grid.x = getGridDim (cols, block.x);
    grid.y = getGridDim (rows, block.y);

    initModelBufferKernel<<<grid, block>>>(intr.cx, intr.cy, intr.fx, intr.fy, rows, cols, depthCutOff, model_buffer, Rmat, tvec, d_count, vmap, nmap, rgb);
    hipDeviceSynchronize();
    cudaCheckError();
    hipMemcpy(h_count, d_count, sizeof(int), hipMemcpyDeviceToHost);
}

// __global__ void kernelCodeKernel(float *result)
// {
//     int index = threadIdx.x+blockIdx.x*blockDim.x;
//     atomicAdd(result, 1.0f);
    
// }
// void kernelCode(){

//     float h_result, *d_result;
//     hipMalloc((void **)&d_result, sizeof(float));
//     h_result = 0.0f;
//     hipMemcpy(d_result, &h_result, sizeof(float), hipMemcpyHostToDevice);

//     int rows, cols;
//     rows = 640;
//     cols = 480;
//     dim3 block(32, 8);
//     dim3 grid(getGridDim(cols, block.x), getGridDim(rows, block.y));

//     kernelCodeKernel<<<grid, block>>>(d_result);
//     hipDeviceSynchronize();
//     hipMemcpy(&h_result, d_result, sizeof(float), hipMemcpyDeviceToHost);
//     std::cout<< "result = " << h_result << std::endl;
// }

__global__ void computeNmapKernel(int rows, int cols, const PtrStep<float> vmap, PtrStep<float> nmap)
{
    int u = threadIdx.x + blockIdx.x * blockDim.x;
    int v = threadIdx.y + blockIdx.y * blockDim.y;

    if (u >= cols || v >= rows)
        return;

    if (u == cols - 1 || v == rows - 1)
    {
        nmap.ptr (v)[u] = __int_as_float(0x7fffffff); /*HIP_NAN_F*/
        return;
    }

    float3 v00, v01, v10;
    v00.x = vmap.ptr (v  )[u];
    v01.x = vmap.ptr (v  )[u + 1];
    v10.x = vmap.ptr (v + 1)[u];

    if (!isnan (v00.x) && !isnan (v01.x) && !isnan (v10.x))
    {
        v00.y = vmap.ptr (v + rows)[u];
        v01.y = vmap.ptr (v + rows)[u + 1];
        v10.y = vmap.ptr (v + 1 + rows)[u];

        v00.z = vmap.ptr (v + 2 * rows)[u];
        v01.z = vmap.ptr (v + 2 * rows)[u + 1];
        v10.z = vmap.ptr (v + 1 + 2 * rows)[u];

        float3 r = normalized (cross (v01 - v00, v10 - v00));

        nmap.ptr (v       )[u] = r.x;
        nmap.ptr (v + rows)[u] = r.y;
        nmap.ptr (v + 2 * rows)[u] = r.z;
    }
    else
        nmap.ptr (v)[u] = __int_as_float(0x7fffffff); /*HIP_NAN_F*/
}

void createNMap(const DeviceArray2D<float>& vmap, DeviceArray2D<float>& nmap)
{
    nmap.create (vmap.rows (), vmap.cols ());

    int rows = vmap.rows () / 3;
    int cols = vmap.cols ();

    dim3 block (32, 8);
    dim3 grid (1, 1, 1);
    grid.x = getGridDim (cols, block.x);
    grid.y = getGridDim (rows, block.y);

    computeNmapKernel<<<grid, block>>>(rows, cols, vmap, nmap);
    cudaSafeCall (hipGetLastError ());
}

__global__ void splatDepthPredictKernel(float cx, float cy, float fx, float fy, int rows, int cols, float* model_buffer, float maxDepth, float confThreshold, int time,  int maxTime,  int timeDelta, const mat33 Rmat_inv, const float3 tvec_inv, float* color_dst, PtrStepSz<float> vmap_dst, PtrStepSz<float> nmap_dst, PtrStepSz<unsigned int> time_dst, int count)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int rows_mb, cols_mb;
    rows_mb = cols_mb = 3072;
    int vp_w, vp_h;
    vp_w = cols;
    vp_h = rows;
    
    if (i >= cols_mb * rows_mb)
        return;
    if (i >= count)
        return;
    if ((model_buffer[i] == 0) && (model_buffer[i + rows_mb*cols_mb] == 0) && (model_buffer[i + 2*rows_mb*cols_mb] == 0)) 
        return;


    // float4 vsrc = make_float4(__int_as_float(0x7fffffff), __int_as_float(0x7fffffff), __int_as_float(0x7fffffff), __int_as_float(0x7fffffff));
    // float4 nsrc = make_float4(__int_as_float(0x7fffffff), __int_as_float(0x7fffffff), __int_as_float(0x7fffffff), __int_as_float(0x7fffffff));

    float4 vsrc = make_float4(0, 0, 0, 0);
    float4 nsrc = make_float4(0, 0, 0, 0);

    //reading vertex and conf
    vsrc.x = model_buffer[i];
    vsrc.y = model_buffer[i + rows_mb*cols_mb];
    vsrc.z = model_buffer[i + 2*rows_mb*cols_mb];
    vsrc.w = model_buffer[i + 3*rows_mb*cols_mb];

    //reading normal and radius
    nsrc.x = model_buffer[i+8*rows_mb*cols_mb];
    nsrc.y = model_buffer[i+9*rows_mb*cols_mb];
    nsrc.z = model_buffer[i+10*rows_mb*cols_mb];
    nsrc.w = model_buffer[i+11*rows_mb*cols_mb];

    // printf("%f\n", model_buffer[i + rows_mb*cols_mb]);

    //reading color
    float c;
    c = model_buffer[i+4*rows_mb*cols_mb]; //x

    //reading time
    unsigned int t; //vcolor.w
    t = (unsigned int)model_buffer[i+7*rows_mb*cols_mb];


    if (isnan (vsrc.x) || isnan(vsrc.y) || isnan(vsrc.z))
        return;
    if (isnan (nsrc.x) || isnan(nsrc.y) || isnan(nsrc.z))
        return;

    float3 v_ = make_float3(__int_as_float(0x7fffffff), __int_as_float(0x7fffffff), __int_as_float(0x7fffffff));
    float3 n_ = make_float3(__int_as_float(0x7fffffff), __int_as_float(0x7fffffff), __int_as_float(0x7fffffff));
    float3 vsrc_xyz = make_float3(vsrc.x,vsrc.y,vsrc.z);
    float3 nsrc_xyz = make_float3(nsrc.x,nsrc.y,nsrc.z);


    v_ = Rmat_inv * vsrc_xyz + tvec_inv;
    n_ = Rmat_inv * nsrc_xyz;
    n_ = normalized(n_);

    // if(v_.z > maxDepth || v_.z < 0 || vsrc.w < confThreshold || time - t > timeDelta || t > maxTime)
    //     return;
    
    if (isnan (v_.x) || isnan(v_.y) || isnan(v_.z))
        return;
    if (isnan (n_.x) || isnan(n_.y) || isnan(n_.z))
        return;

    //to compute x,y cords (gl_fragcords)
    //TO DO need to normalize v_ 
    float3 fc;
    fc = projectPoint(v_, rows, cols, cx, cy, fx, fy, maxDepth);
    fc.x = fc.x * 0.5f + 0.5f; 
    fc.y = fc.y * 0.5f + 0.5f; 
    fc.x = fc.x * vp_w;
    fc.y = fc.y * vp_h;

    int x, y;
    x = (int)fc.x;
    y = (int)fc.y;
    // printf("x = %d y = %d\n", x, y);

    if (x < 0 || x > cols || y < 0 || y > rows)
        return;

    float3 l = make_float3(__int_as_float(0x7fffffff), __int_as_float(0x7fffffff), __int_as_float(0x7fffffff));
    float3 cp = make_float3(__int_as_float(0x7fffffff), __int_as_float(0x7fffffff), __int_as_float(0x7fffffff));

    l.x = (fc.x - cx)/fx;
    l.y = (fc.y - cy)/fy;
    l.z = 1;
    l = normalized(l);

    float coeff;
    coeff = dot(v_, n_) / dot(l, n_);
    cp.x = l.x * coeff;
    cp.y = l.y * coeff;
    cp.z = l.z * coeff;

    float sqrRad = pow(nsrc.w, 2);
    float3 diff;
    diff.x = cp.x - vsrc.x;
    diff.y = cp.y - vsrc.y;
    diff.z = cp.z - vsrc.z;

    if (dot(diff,diff) > sqrRad)
    {
        return;
    }

    float3 dc; 
    dc = decodeColor(c);

    //writing color TO DO UPDATE disableing color decoding
    color_dst[y*cols*4 + x*4 + 0] = dc.x;
    color_dst[y*cols*4 + x*4 + 1] = dc.y;
    color_dst[y*cols*4 + x*4 + 2] = dc.z;
    color_dst[y*cols*4 + x*4 + 3] = 1;
    // color_dst[y*cols*4 + x*4 + 0] = model_buffer[i+4*rows_mb*cols_mb];
    // color_dst[y*cols*4 + x*4 + 1] = model_buffer[i+5*rows_mb*cols_mb];
    // color_dst[y*cols*4 + x*4 + 2] = model_buffer[i+6*rows_mb*cols_mb];
    // color_dst[y*cols*4 + x*4 + 3] = 1;

    //writing vertex and conf

    float3 sp = make_float3(0,0,0);
    sp.x = (fc.x - cx)*cp.z*(1/fx);
    sp.y = (fc.y - cy)*cp.z*(1/fy);
    sp.z = cp.z;

    if ((sp.z < 0) || (sp.z > maxDepth))
    {
        return;
    }
    // if ((sp.x < 0) || (sp.x > cols))
    // {
    //     return;
    // }
    // if ((sp.y < 0) || (sp.y > rows))
    // {
    //     return;
    // }

    vmap_dst.ptr(y)[x] = sp.x;
    vmap_dst.ptr(y + rows)[x] = sp.y;
    vmap_dst.ptr(y + rows * 2)[x] = sp.z;
    vmap_dst.ptr(y + rows * 3)[x] = vsrc.w;


    //writing normal and radius
    nmap_dst.ptr(y       )[x] = n_.x;
    nmap_dst.ptr(y + rows)[x] = n_.y;
    nmap_dst.ptr(y + 2 * rows)[x] = n_.z;
    nmap_dst.ptr(y + 3 * rows)[x] = nsrc.w;

    //writing time
    time_dst.ptr(y)[x] = t;

}

void splatDepthPredict(const CameraModel& intr, int rows, int cols, DeviceArray<float>& model_buffer, float depthCutoff, float confThreshold, int time, int maxTime, int timeDelta, const mat33 Rmat_inv, const float3 tvec_inv, int count, DeviceArray<float>& color_dst, DeviceArray2D<float>& vmap_dst, DeviceArray2D<float>& nmap_dst, DeviceArray2D<unsigned int>& time_dst)
{
    int blocksize = 32*8;
    int numblocks = (count + blocksize - 1)/ blocksize;

    float fx = intr.fx, cx = intr.cx;
    float fy = intr.fy, cy = intr.cy;

    float* vertices_splat = new float[rows*cols*4];

    vmap_dst.upload(&vertices_splat[0], sizeof(float)*cols, 4*rows, cols);

    color_dst.upload(&vertices_splat[0], rows*4*cols);

    nmap_dst.upload(&vertices_splat[0], sizeof(float)*cols, 4*rows, cols);
    
    time_dst.upload(&vertices_splat[0], sizeof(float)*cols, rows, cols);

    delete[] vertices_splat;

    // std::cout<<"cx = "<<cx<<": cy = "<<cy<<": fx = "<<fx<<": fy = "<<fy<<": rows = "<<rows<<": cols = "<<cols;
    splatDepthPredictKernel<<<numblocks, blocksize>>>(cx, cy, fx, fy, rows, cols, model_buffer, depthCutoff, confThreshold, time, maxTime, timeDelta, Rmat_inv, tvec_inv, color_dst, vmap_dst, nmap_dst, time_dst, count);
    cudaCheckError();

}

__global__ void tranformMapsKernel(int rows, int cols, const PtrStep<float> vmap_src, const PtrStep<float> nmap_src,
                                   const mat33 Rmat, const float3 tvec, PtrStepSz<float> vmap_dst, PtrStep<float> nmap_dst)
{
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;

    if (x < cols && y < rows)
    {
        //vertexes
        float3 vsrc, vdst = make_float3 (__int_as_float(0x7fffffff), __int_as_float(0x7fffffff), __int_as_float(0x7fffffff));
        vsrc.x = vmap_src.ptr (y)[x];

        if (!isnan (vsrc.x))
        {
            vsrc.y = vmap_src.ptr (y + rows)[x];
            vsrc.z = vmap_src.ptr (y + 2 * rows)[x];

            vdst = Rmat * vsrc + tvec;

            vmap_dst.ptr (y + rows)[x] = vdst.y;
            vmap_dst.ptr (y + 2 * rows)[x] = vdst.z;
        }

        vmap_dst.ptr (y)[x] = vdst.x;

        //normals
        float3 nsrc, ndst = make_float3 (__int_as_float(0x7fffffff), __int_as_float(0x7fffffff), __int_as_float(0x7fffffff));
        nsrc.x = nmap_src.ptr (y)[x];

        if (!isnan (nsrc.x))
        {
            nsrc.y = nmap_src.ptr (y + rows)[x];
            nsrc.z = nmap_src.ptr (y + 2 * rows)[x];

            ndst = Rmat * nsrc;

            nmap_dst.ptr (y + rows)[x] = ndst.y;
            nmap_dst.ptr (y + 2 * rows)[x] = ndst.z;
        }

        nmap_dst.ptr (y)[x] = ndst.x;
    }
}

void tranformMaps(const DeviceArray2D<float>& vmap_src,
                  const DeviceArray2D<float>& nmap_src,
                  const mat33& Rmat, const float3& tvec,
                  DeviceArray2D<float>& vmap_dst, DeviceArray2D<float>& nmap_dst)
{
    int cols = vmap_src.cols();
    int rows = vmap_src.rows() / 3;

    vmap_dst.create(rows * 3, cols);
    nmap_dst.create(rows * 3, cols);

    dim3 block(32, 8);
    dim3 grid(1, 1, 1);
    grid.x = getGridDim(cols, block.x);
    grid.y = getGridDim(rows, block.y);

    tranformMapsKernel<<<grid, block>>>(rows, cols, vmap_src, nmap_src, Rmat, tvec, vmap_dst, nmap_dst);
    cudaSafeCall(hipGetLastError());
}

__global__ void copyMapsKernel(int rows, int cols, const float * vmap_src, const float * nmap_src,
                               PtrStepSz<float> vmap_dst, PtrStep<float> nmap_dst)
{
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;

    if (x < cols && y < rows)
    {
        //vertexes
        float3 vsrc, vdst = make_float3 (__int_as_float(0x7fffffff), __int_as_float(0x7fffffff), __int_as_float(0x7fffffff));

        vsrc.x = vmap_src[y * cols * 4 + (x * 4) + 0];
        vsrc.y = vmap_src[y * cols * 4 + (x * 4) + 1];
        vsrc.z = vmap_src[y * cols * 4 + (x * 4) + 2];

        if(!(vsrc.z == 0))
        {
            vdst = vsrc;
        }

        vmap_dst.ptr (y)[x] = vdst.x;
        vmap_dst.ptr (y + rows)[x] = vdst.y;
        vmap_dst.ptr (y + 2 * rows)[x] = vdst.z;

        //normals
        float3 nsrc, ndst = make_float3 (__int_as_float(0x7fffffff), __int_as_float(0x7fffffff), __int_as_float(0x7fffffff));

        nsrc.x = nmap_src[y * cols * 4 + (x * 4) + 0];
        nsrc.y = nmap_src[y * cols * 4 + (x * 4) + 1];
        nsrc.z = nmap_src[y * cols * 4 + (x * 4) + 2];

        if(!(vsrc.z == 0))
        {
            ndst = nsrc;
        }

        nmap_dst.ptr (y)[x] = ndst.x;
        nmap_dst.ptr (y + rows)[x] = ndst.y;
        nmap_dst.ptr (y + 2 * rows)[x] = ndst.z;
    }
}

void copyMaps(const DeviceArray<float>& vmap_src,
              const DeviceArray<float>& nmap_src,
              DeviceArray2D<float>& vmap_dst,
              DeviceArray2D<float>& nmap_dst)
{
    int cols = vmap_dst.cols();
    int rows = vmap_dst.rows() / 3;

    vmap_dst.create(rows * 3, cols);
    nmap_dst.create(rows * 3, cols);

    dim3 block(32, 8);
    dim3 grid(1, 1, 1);
    grid.x = getGridDim(cols, block.x);
    grid.y = getGridDim(rows, block.y);

    copyMapsKernel<<<grid, block>>>(rows, cols, vmap_src, nmap_src, vmap_dst, nmap_dst);
    cudaSafeCall(hipGetLastError());
}



__global__ void copyMapsKernel2D_2_2D(int rows, int cols, PtrStepSz<float> vmap_src, PtrStep<float> nmap_src,
                               PtrStepSz<float> vmap_dst, PtrStep<float> nmap_dst)
{
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;

    if (x < cols && y < rows)
    {
        // vertexes
        float3 vsrc, vdst = make_float3 (__int_as_float(0x7fffffff), __int_as_float(0x7fffffff), __int_as_float(0x7fffffff));

        vsrc.x = vmap_src.ptr(y)[x];
        vsrc.y = vmap_src.ptr(y+rows)[x];
        vsrc.z = vmap_src.ptr(y+2*rows)[x];

        if(!(vsrc.z == 0))
        {
            vdst = vsrc;
        }

        vmap_dst.ptr (y)[x] = vdst.x;
        vmap_dst.ptr (y + rows)[x] = vdst.y;
        vmap_dst.ptr (y + 2 * rows)[x] = vdst.z;

        //normals
        float3 nsrc, ndst = make_float3 (__int_as_float(0x7fffffff), __int_as_float(0x7fffffff), __int_as_float(0x7fffffff));

        nsrc.x = nmap_src.ptr(y)[x];
        nsrc.y = nmap_src.ptr(y+rows)[x];
        nsrc.z = nmap_src.ptr(y+2*rows)[x];

        if(!(vsrc.z == 0))
        {
            ndst = nsrc;
        }
        nmap_dst.ptr (y)[x] = ndst.x;
        nmap_dst.ptr (y + rows)[x] = ndst.y;
        nmap_dst.ptr (y + 2 * rows)[x] = ndst.z;
    }
}

void copyMaps(const DeviceArray2D<float>& vmap_src,
              const DeviceArray2D<float>& nmap_src,
              DeviceArray2D<float>& vmap_dst,
              DeviceArray2D<float>& nmap_dst)
{
    int cols = vmap_dst.cols();
    int rows = vmap_dst.rows() / 3;

    vmap_dst.create(rows * 3, cols);
    nmap_dst.create(rows * 3, cols);

    dim3 block(32, 8);
    dim3 grid(1, 1, 1);
    grid.x = getGridDim(cols, block.x);
    grid.y = getGridDim(rows, block.y);

    copyMapsKernel2D_2_2D<<<grid, block>>>(rows, cols, vmap_src, nmap_src, vmap_dst, nmap_dst);
    cudaSafeCall(hipGetLastError());
}

__global__ void copyMapsKernel2D_2_1D(int rows, int cols, PtrStepSz<float> vmap_src, PtrStep<float> nmap_src,
                                 float * vmap_dst,   float * nmap_dst)
{
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;

    if (x < cols && y < rows)
    {
        //vertexes
        float4 vsrc, vdst = make_float4 (__int_as_float(0x7fffffff), __int_as_float(0x7fffffff), __int_as_float(0x7fffffff),  __int_as_float(0x7fffffff));

        vsrc.x = vmap_src.ptr (y)[x];
        vsrc.y = vmap_src.ptr (y + rows)[x];
        vsrc.z = vmap_src.ptr (y + 2 * rows)[x];
        vsrc.w = vmap_src.ptr (y + 3 * rows)[x];


        if(!(vsrc.z == 0))
        {
            vdst = vsrc;
        }

        vmap_dst[y * cols * 4 + (x * 4) + 0] = vdst.x;
        vmap_dst[y * cols * 4 + (x * 4) + 1] = vdst.y;
        vmap_dst[y * cols * 4 + (x * 4) + 2] = vdst.z;
        vmap_dst[y * cols * 4 + (x * 4) + 3] = vdst.w;


        //normals
        float4 nsrc, ndst = make_float4 (__int_as_float(0x7fffffff), __int_as_float(0x7fffffff), __int_as_float(0x7fffffff),  __int_as_float(0x7fffffff));

        nsrc.x = nmap_src.ptr (y)[x] ;
        nsrc.y = nmap_src.ptr (y + rows)[x] ;
        nsrc.z = nmap_src.ptr (y + 2 * rows)[x] ;
        nsrc.w = nmap_src.ptr (y + 3 * rows)[x] ;


        if(!(vsrc.z == 0))
        {
            ndst = nsrc;
        }

        nmap_dst[y * cols * 4 + (x * 4) + 0]= ndst.x;
        nmap_dst[y * cols * 4 + (x * 4) + 1]= ndst.y;
        nmap_dst[y * cols * 4 + (x * 4) + 2]= ndst.z;
        nmap_dst[y * cols * 4 + (x * 4) + 3]= ndst.w;

    }
}

void copyMaps(const DeviceArray2D<float>& vmap_src,
              const DeviceArray2D<float>& nmap_src,
              DeviceArray<float>& vmap_dst,
              DeviceArray<float>& nmap_dst)
{
    int rows = vmap_src.rows() / 4;
    int cols = vmap_src.cols();

    vmap_dst.create(rows * 4 * cols);
    nmap_dst.create(rows * 4 * cols);

    dim3 block(32, 8);
    dim3 grid(1, 1, 1);
    grid.x = getGridDim(cols, block.x);
    grid.y = getGridDim(rows, block.y);


    copyMapsKernel2D_2_1D<<<grid, block>>>(rows, cols, vmap_src, nmap_src, vmap_dst, nmap_dst);
    cudaSafeCall(hipGetLastError());
}

__global__ void copyDMapsKernel2D_2_2D(int rows, int cols, PtrStepSz<float> dmap_src, PtrStepSz<float> dmap_dst)
{
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;

    if (x < cols && y < rows)
    {
        dmap_dst.ptr(y)[x] = dmap_src.ptr(y)[x];
    }
}

void copyDMaps(const DeviceArray2D<float>& dmap_src,
              DeviceArray2D<float>& dmap_dst)
{
    int cols = dmap_src.cols();
    int rows = dmap_src.rows();


    dim3 block(32, 8);
    dim3 grid(1, 1, 1);
    grid.x = getGridDim(cols, block.x);
    grid.y = getGridDim(rows, block.y);

    copyDMapsKernel2D_2_2D<<<grid, block>>>(rows, cols, dmap_src, dmap_dst);
    cudaSafeCall(hipGetLastError());
}

__global__ void pyrDownKernelGaussF(const PtrStepSz<float> src, PtrStepSz<float> dst, float * gaussKernel)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= dst.cols || y >= dst.rows)
        return;

    const int D = 5;

    float center = src.ptr (2 * y)[2 * x];

    int tx = min (2 * x - D / 2 + D, src.cols - 1);
    int ty = min (2 * y - D / 2 + D, src.rows - 1);
    int cy = max (0, 2 * y - D / 2);

    float sum = 0;
    int count = 0;

    for (; cy < ty; ++cy)
    {
        for (int cx = max (0, 2 * x - D / 2); cx < tx; ++cx)
        {
            if(!isnan(src.ptr (cy)[cx]))
            {
                sum += src.ptr (cy)[cx] * gaussKernel[(ty - cy - 1) * 5 + (tx - cx - 1)];
                count += gaussKernel[(ty - cy - 1) * 5 + (tx - cx - 1)];
            }
        }
    }
    dst.ptr (y)[x] = (float)(sum / (float)count);
}

template<bool normalize>
__global__ void resizeMapKernel(int drows, int dcols, int srows, const PtrStep<float> input, PtrStep<float> output)
{
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;

    if (x >= dcols || y >= drows)
        return;

    const float qnan = __int_as_float(0x7fffffff);

    int xs = x * 2;
    int ys = y * 2;

    float x00 = input.ptr (ys + 0)[xs + 0];
    float x01 = input.ptr (ys + 0)[xs + 1];
    float x10 = input.ptr (ys + 1)[xs + 0];
    float x11 = input.ptr (ys + 1)[xs + 1];

    if (isnan (x00) || isnan (x01) || isnan (x10) || isnan (x11))
    {
        output.ptr (y)[x] = qnan;
        return;
    }
    else
    {
        float3 n;

        n.x = (x00 + x01 + x10 + x11) / 4;

        float y00 = input.ptr (ys + srows + 0)[xs + 0];
        float y01 = input.ptr (ys + srows + 0)[xs + 1];
        float y10 = input.ptr (ys + srows + 1)[xs + 0];
        float y11 = input.ptr (ys + srows + 1)[xs + 1];

        n.y = (y00 + y01 + y10 + y11) / 4;

        float z00 = input.ptr (ys + 2 * srows + 0)[xs + 0];
        float z01 = input.ptr (ys + 2 * srows + 0)[xs + 1];
        float z10 = input.ptr (ys + 2 * srows + 1)[xs + 0];
        float z11 = input.ptr (ys + 2 * srows + 1)[xs + 1];

        n.z = (z00 + z01 + z10 + z11) / 4;

        if (normalize)
            n = normalized (n);

        output.ptr (y        )[x] = n.x;
        output.ptr (y + drows)[x] = n.y;
        output.ptr (y + 2 * drows)[x] = n.z;
    }
}

template<bool normalize>
void resizeMap(const DeviceArray2D<float>& input, DeviceArray2D<float>& output)
{
    int in_cols = input.cols ();
    int in_rows = input.rows () / 3;

    int out_cols = in_cols / 2;
    int out_rows = in_rows / 2;

    output.create (out_rows * 3, out_cols);

    dim3 block (32, 8);
    dim3 grid (getGridDim (out_cols, block.x), getGridDim (out_rows, block.y));
    resizeMapKernel<normalize><< < grid, block>>>(out_rows, out_cols, in_rows, input, output);
    cudaCheckError();
    cudaSafeCall (hipDeviceSynchronize ());
}

void resizeVMap(const DeviceArray2D<float>& input, DeviceArray2D<float>& output)
{
    resizeMap<false>(input, output);
}

void resizeNMap(const DeviceArray2D<float>& input, DeviceArray2D<float>& output)
{
    resizeMap<true>(input, output);
}

//FIXME Remove
/*
void launch_kernel(float4 *pos, unsigned int mesh_width,
                   unsigned int mesh_height, float time)
{
    // execute the kernel
    dim3 block(8, 8, 1);
    dim3 grid(mesh_width / block.x, mesh_height / block.y, 1);
    simple_vbo_kernel<<< grid, block>>>(pos, mesh_width, mesh_height, time);
}*/

//FIXME Remove
/*
__global__ void testKernel(hipSurfaceObject_t tex)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= 960 || y >= 540)
        return;

    / *
    const int D = 5;

    float center = src.ptr (2 * y)[2 * x];

    int tx = min (2 * x - D / 2 + D, src.cols - 1);
    int ty = min (2 * y - D / 2 + D, src.rows - 1);
    int cy = max (0, 2 * y - D / 2);

    float sum = 0;
    int count = 0;

    for (; cy < ty; ++cy)
    {
        for (int cx = max (0, 2 * x - D / 2); cx < tx; ++cx)
        {
            if(!isnan(src.ptr (cy)[cx]))
            {
                sum += src.ptr (cy)[cx] * gaussKernel[(ty - cy - 1) * 5 + (tx - cx - 1)];
                count += gaussKernel[(ty - cy - 1) * 5 + (tx - cx - 1)];
            }
        }
    }* /
    //dst.ptr (y)[x] = (float)(sum / (float)count);
    //data[y * 960 + x] = x / 960.0;
    //data[8] = 0.4;
    float1 test = make_float1(0.99);
    surf2Dwrite(test, tex, x*sizeof(float1), y);
}

//FIXME Remove
void testCuda(hipSurfaceObject_t surface)//(float* data)
{
    //dst.create (src.rows () / 2, src.cols () / 2);

    dim3 block (32, 8);
    dim3 grid (getGridDim (960, block.x), getGridDim (540, block.y));

    testKernel<<<grid, block>>>(surface);
    cudaCheckError();
}*/

void pyrDownGaussF(const DeviceArray2D<float>& src, DeviceArray2D<float> & dst)
{
    dst.create (src.rows () / 2, src.cols () / 2);

    dim3 block (32, 8);
    dim3 grid (getGridDim (dst.cols (), block.x), getGridDim (dst.rows (), block.y));

    const float gaussKernel[25] = {1, 4, 6, 4, 1,
                    4, 16, 24, 16, 4,
                    6, 24, 36, 24, 6,
                    4, 16, 24, 16, 4,
                    1, 4, 6, 4, 1};

    float * gauss_cuda;

    cudaSafeCall(hipMalloc((void**) &gauss_cuda, sizeof(float) * 25));
    cudaSafeCall(hipMemcpy(gauss_cuda, &gaussKernel[0], sizeof(float) * 25, hipMemcpyHostToDevice));

    pyrDownKernelGaussF<<<grid, block>>>(src, dst, gauss_cuda);
    cudaCheckError();

    hipFree(gauss_cuda);
}

__global__ void pyrDownKernelIntensityGauss(const PtrStepSz<unsigned char> src, PtrStepSz<unsigned char> dst, float * gaussKernel)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= dst.cols || y >= dst.rows)
        return;

    const int D = 5;

    int center = src.ptr (2 * y)[2 * x];

    int tx = min (2 * x - D / 2 + D, src.cols - 1);
    int ty = min (2 * y - D / 2 + D, src.rows - 1);
    int cy = max (0, 2 * y - D / 2);

    float sum = 0;
    int count = 0;

    for (; cy < ty; ++cy)
        for (int cx = max (0, 2 * x - D / 2); cx < tx; ++cx)
        {
            //This might not be right, but it stops incomplete model images from making up colors
            if(src.ptr (cy)[cx] > 0)
            {
                sum += src.ptr (cy)[cx] * gaussKernel[(ty - cy - 1) * 5 + (tx - cx - 1)];
                count += gaussKernel[(ty - cy - 1) * 5 + (tx - cx - 1)];
            }
        }
    dst.ptr (y)[x] = (sum / (float)count);
}

void pyrDownUcharGauss(const DeviceArray2D<unsigned char>& src, DeviceArray2D<unsigned char> & dst)
{
    dst.create (src.rows () / 2, src.cols () / 2);

    dim3 block (32, 8);
    dim3 grid (getGridDim (dst.cols (), block.x), getGridDim (dst.rows (), block.y));

    const float gaussKernel[25] = {1, 4, 6, 4, 1,
                    4, 16, 24, 16, 4,
                    6, 24, 36, 24, 6,
                    4, 16, 24, 16, 4,
                    1, 4, 6, 4, 1};

    float * gauss_cuda;

    hipMalloc((void**) &gauss_cuda, sizeof(float) * 25);
    hipMemcpy(gauss_cuda, &gaussKernel[0], sizeof(float) * 25, hipMemcpyHostToDevice);

    pyrDownKernelIntensityGauss<<<grid, block>>>(src, dst, gauss_cuda);
    cudaCheckError();

    hipFree(gauss_cuda);
}

/*void pyrDown2(const DeviceArray2D<unsigned char> & src, DeviceArray2D<unsigned char> & dst)
{
    dst.create (src.rows () / 2, src.cols () / 2);

    dim3 block (32, 8);
    dim3 grid (getGridDim (dst.cols (), block.x), getGridDim (dst.rows (), block.y));

    //pyrDownUcharGauss<<<grid, block>>>(src, dst);
    pyrDownUcharGauss()
    cudaCheckError();
}*/

__global__ void verticesToDepthKernel(const float * vmap_src, PtrStepSz<float> dst, float cutOff)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= dst.cols || y >= dst.rows)
        return;

    float z = vmap_src[y * dst.cols * 4 + (x * 4) + 2];

    dst.ptr(y)[x] = z > cutOff || z <= 0 ? __int_as_float(0x7fffffff)/*HIP_NAN_F*/ : z;
}

void verticesToDepth(DeviceArray<float>& vmap_src, DeviceArray2D<float> & dst, float cutOff)
{
    dim3 block (32, 8);
    dim3 grid (getGridDim (dst.cols (), block.x), getGridDim (dst.rows (), block.y));

    verticesToDepthKernel<<<grid, block>>>(vmap_src, dst, cutOff);
    cudaCheckError();
}

texture<uchar4, 2, hipReadModeElementType> inTex;

__global__ void bgr2IntensityKernel(PtrStepSz<unsigned char> dst)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= dst.cols || y >= dst.rows)
        return;

    uchar4 src = tex2D(inTex, x, y);

    int value = (float)src.x * 0.114f + (float)src.y * 0.299f + (float)src.z * 0.587f;

    printf("%d\n", value);

    dst.ptr (y)[x] = value;
}

void imageBGRToIntensity(hipArray * cuArr, DeviceArray2D<unsigned char> & dst)
{
    dim3 block (32, 8);
    dim3 grid (getGridDim (dst.cols (), block.x), getGridDim (dst.rows (), block.y));

    cudaSafeCall(hipBindTextureToArray(inTex, cuArr));

    bgr2IntensityKernel<<<grid, block>>>(dst);

    cudaCheckError();

    cudaSafeCall(hipUnbindTexture(inTex));
}

__global__ void bgr2IntensityKernelDMC3(int rows, int cols, float * rgb_src, PtrStepSz<unsigned char> rgb_dst)
{
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    if (x < cols && y < rows)
    {
        float3 vsrc, vdst = make_float3 (__int_as_float(0x7fffffff), __int_as_float(0x7fffffff), __int_as_float(0x7fffffff));
        vsrc.x = rgb_src[y * cols * 3 + (x * 3) + 0];
        vsrc.y = rgb_src[y * cols * 3 + (x * 3) + 1];
        vsrc.z = rgb_src[y * cols * 3 + (x * 3) + 2];
        int value = (float)vsrc.x * 0.114f + (float)vsrc.y * 0.299f + (float)vsrc.z * 0.587f;
        rgb_dst.ptr(y)[x] = value;

    }
}

__global__ void bgr2IntensityKernelDMC4(int rows, int cols, float * rgb_src, PtrStepSz<unsigned char> rgb_dst)
{
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    if (x < cols && y < rows)
    {
        float3 vsrc, vdst = make_float3 (__int_as_float(0x7fffffff), __int_as_float(0x7fffffff), __int_as_float(0x7fffffff));
        vsrc.x = rgb_src[y * cols * 4 + (x * 4) + 0];
        vsrc.y = rgb_src[y * cols * 4 + (x * 4) + 1];
        vsrc.z = rgb_src[y * cols * 4 + (x * 4) + 2];
        int value = (float)vsrc.x * 0.114f + (float)vsrc.y * 0.299f + (float)vsrc.z * 0.587f;
        rgb_dst.ptr(y)[x] = value;

    }
}

void imageBGRToIntensityDM(DeviceArray<float>& rgb_src, bool c3, DeviceArray2D<unsigned char>& rgb_dst)
{
    dim3 block (32, 8);
    dim3 grid (getGridDim (rgb_dst.cols(), block.x), getGridDim (rgb_dst.rows(), block.y));
    int rows = rgb_dst.rows(); // TO DO CHANGED changed / 3
    int cols = rgb_dst.cols();
    if (c3)
        bgr2IntensityKernelDMC3<<<grid, block>>>(rows, cols, rgb_src, rgb_dst);
    else
        bgr2IntensityKernelDMC4<<<grid, block>>>(rows, cols, rgb_src, rgb_dst);

    cudaCheckError();

}

__constant__ float gsobel_x3x3[9];
__constant__ float gsobel_y3x3[9];

__global__ void applyKernel(const PtrStepSz<unsigned char> src, PtrStep<short> dx, PtrStep<short> dy)
{

  int x = threadIdx.x + blockIdx.x * blockDim.x;
  int y = threadIdx.y + blockIdx.y * blockDim.y;

  if(x >= src.cols || y >= src.rows)
    return;

  float dxVal = 0;
  float dyVal = 0;

  int kernelIndex = 8;
  for(int j = max(y - 1, 0); j <= min(y + 1, src.rows - 1); j++)
  {
      for(int i = max(x - 1, 0); i <= min(x + 1, src.cols - 1); i++)
      {
          dxVal += (float)src.ptr(j)[i] * gsobel_x3x3[kernelIndex];
          dyVal += (float)src.ptr(j)[i] * gsobel_y3x3[kernelIndex];
          --kernelIndex;
      }
  }

  dx.ptr(y)[x] = dxVal;
  dy.ptr(y)[x] = dyVal;
}

void computeDerivativeImages(DeviceArray2D<unsigned char>& src, DeviceArray2D<short>& dx, DeviceArray2D<short>& dy)
{
    static bool once = false;

    if(!once)
    {
        float gsx3x3[9] = {0.52201,  0.00000, -0.52201,
                           0.79451, -0.00000, -0.79451,
                           0.52201,  0.00000, -0.52201};

        float gsy3x3[9] = {0.52201, 0.79451, 0.52201,
                           0.00000, 0.00000, 0.00000,
                           -0.52201, -0.79451, -0.52201};

        hipMemcpyToSymbol(HIP_SYMBOL(gsobel_x3x3), gsx3x3, sizeof(float) * 9);
        hipMemcpyToSymbol(HIP_SYMBOL(gsobel_y3x3), gsy3x3, sizeof(float) * 9);

        cudaSafeCall(hipGetLastError());
        cudaSafeCall(hipDeviceSynchronize());

        once = true;
    }

    dim3 block(32, 8);
    dim3 grid(getGridDim (src.cols (), block.x), getGridDim (src.rows (), block.y));

    applyKernel<<<grid, block>>>(src, dx, dy);

    cudaSafeCall(hipGetLastError());
    cudaSafeCall(hipDeviceSynchronize());
}


__global__ void projectPointsKernel(const PtrStepSz<float> depth,
                                    PtrStepSz<float3> cloud,
                                    const float invFx,
                                    const float invFy,
                                    const float cx,
                                    const float cy)
{
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;

    if (x >= depth.cols || y >= depth.rows)
        return;

    float z = depth.ptr(y)[x];

    cloud.ptr(y)[x].x = (float)((x - cx) * z * invFx);
    cloud.ptr(y)[x].y = (float)((y - cy) * z * invFy);
    cloud.ptr(y)[x].z = z;
}

void projectToPointCloud(const DeviceArray2D<float> & depth,
                         const DeviceArray2D<float3> & cloud,
                         CameraModel & intrinsics,
                         const int & level)
{
    dim3 block (32, 8);
    dim3 grid (getGridDim (depth.cols (), block.x), getGridDim (depth.rows (), block.y));

    CameraModel intrinsicsLevel = intrinsics(level);

    projectPointsKernel<<<grid, block>>>(depth, cloud, 1.0f / intrinsicsLevel.fx, 1.0f / intrinsicsLevel.fy, intrinsicsLevel.cx, intrinsicsLevel.cy);
    cudaCheckError();
    cudaSafeCall (hipDeviceSynchronize ());
}


__global__ void predictIndiciesOpenGLKernel(float cx, float cy, float fx, float fy,  int rows, int cols, float maxDepth, float* tinv, float* model_buffer, int time, int timeDelta, PtrStepSz<float> vmap_pi, PtrStepSz<float> ct_pi, PtrStepSz<float> nmap_pi, PtrStepSz<unsigned int> index_pi)
{

    int i = threadIdx.x + blockIdx.x * blockDim.x;

    int rows_mb, cols_mb;
    rows_mb = cols_mb = 3072;
    // int i = y* rows + x;
    float xu = 0;
    float yv = 0;

    if (i >= rows_mb*cols_mb)
        return;

    int vz = model_buffer[i + 2*rows_mb*cols_mb];
    int cw = model_buffer[i+7*rows_mb*cols_mb];
    int vertexId;
    float3 vsrc = make_float3(__int_as_float(0x7fffffff), __int_as_float(0x7fffffff), __int_as_float(0x7fffffff));

    if ((vz < 0 ) || (vz > maxDepth) /*|| (time - cw > timeDelta)*/)
    {
        vsrc.x = -10;
        vsrc.y = -10;
        vertexId = 0;
    }
    else
    {
        float3 v_ = make_float3(__int_as_float(0x7fffffff), __int_as_float(0x7fffffff), __int_as_float(0x7fffffff));
        float3 nsrc = make_float3(__int_as_float(0x7fffffff), __int_as_float(0x7fffffff), __int_as_float(0x7fffffff));
        float3 n_ = make_float3(__int_as_float(0x7fffffff), __int_as_float(0x7fffffff), __int_as_float(0x7fffffff));

        vsrc.x = model_buffer[i];
        vsrc.y = model_buffer[i + rows_mb*cols_mb];
        vsrc.z = model_buffer[i + 2*rows_mb*cols_mb];

        nsrc.x = model_buffer[i+8*rows_mb*cols_mb];
        nsrc.y = model_buffer[i+9*rows_mb*cols_mb];
        nsrc.z = model_buffer[i+10*rows_mb*cols_mb];

        v_.x = tinv[0]*vsrc.x + tinv[1]*vsrc.y + tinv[2]*vsrc.z + tinv[3]*1;
        v_.y = tinv[4]*vsrc.x + tinv[5]*vsrc.y + tinv[6]*vsrc.z + tinv[7]*1;
        v_.z = tinv[8]*vsrc.x + tinv[9]*vsrc.y + tinv[10]*vsrc.z + tinv[11]*1;

        // xu = ((((fx* v_.x) / v_.z) + cx) - (cols * 0.5)) / (cols * 0.5);
        // yv = ((((fy * v_.y) / v_.z) + cy) - (rows * 0.5)) / (rows * 0.5);
        // vertexId = gl_VertexID;
        vertexId = i;

        n_.x = tinv[0]*nsrc.x + tinv[1]*nsrc.y + tinv[2]*nsrc.z;
        n_.y = tinv[4]*nsrc.x + tinv[5]*nsrc.y + tinv[6]*nsrc.z;
        n_.z = tinv[8]*nsrc.x + tinv[9]*nsrc.y + tinv[10]*nsrc.z;
        n_ = normalized(n_);


        float3 fc;
        fc = projectPoint(v_, rows, cols, cx, cy, fx, fy, maxDepth);
        fc.x = fc.x * 0.5f + 0.5f; 
        fc.y = fc.y * 0.5f + 0.5f; 
        fc.x = fc.x * cols;
        fc.y = fc.y * rows;
        int x = fc.x, y = fc.y;

        if (x < 0 || x > cols || y < 0 || y > rows)
            return;


        // printf("x = %d y = %d\n", x, y);
        vmap_pi.ptr(y)[x] = v_.x;
        vmap_pi.ptr(y + rows)[x] = v_.y;
        vmap_pi.ptr(y + rows * 2)[x] = v_.z;
        vmap_pi.ptr(y + rows * 3)[x] = model_buffer[i + 3*rows_mb*cols_mb];

        ct_pi.ptr(y)[x] = model_buffer[i+4*rows_mb*cols_mb];
        ct_pi.ptr(y + rows)[x] = model_buffer[i+5*rows_mb*cols_mb];
        ct_pi.ptr(y + rows * 2)[x] = model_buffer[i+6*rows_mb*cols_mb];
        ct_pi.ptr(y + rows * 3)[x] = model_buffer[i+7*rows_mb*cols_mb];

        nmap_pi.ptr(y)[x] = n_.x;
        nmap_pi.ptr(y + rows)[x] = n_.y;
        nmap_pi.ptr(y + rows * 2)[x] = n_.z;
        nmap_pi.ptr(y + rows * 3)[x] = model_buffer[i + 11*rows_mb*cols_mb];

        index_pi.ptr(y)[x] = i;
    }
}

void predictIndiciesOpenGL(const CameraModel& intr, int rows, int cols, float maxDepth, float* pose_inv, DeviceArray<float>& model_buffer, int time, DeviceArray2D<float>& vmap_pi, DeviceArray2D<float>& ct_pi, DeviceArray2D<float>& nmap_pi, DeviceArray2D<unsigned int>& index_pi, int count)
{
    int blocksize = 32*8;
    int numblocks = (count + blocksize - 1)/ blocksize;
    int timeDelta = 200;

    float* vertices = new float[rows*cols*4];
    memset(&vertices[0], 0, rows*cols*4);

    vmap_pi.create(rows*4, cols); 
    vmap_pi.upload(&vertices[0], sizeof(float)*cols, 4*rows, cols);

    ct_pi.create(rows*4, cols);
    ct_pi.upload(&vertices[0], sizeof(float)*cols, 4*rows, cols);

    nmap_pi.create(rows*4, cols);
    nmap_pi.upload(&vertices[0], sizeof(float)*cols, 4*rows, cols);
    
    index_pi.create(rows,cols);
    index_pi.upload(&vertices[0], sizeof(float)*cols, rows, cols);

    delete[] vertices;

    float fx = intr.fx, cx = intr.cx;
    float fy = intr.fy, cy = intr.cy;

    float * tinv;
    cudaSafeCall(hipMalloc((void**) &tinv, sizeof(float) * 16));
    cudaSafeCall(hipMemcpy(tinv, pose_inv, sizeof(float) * 16, hipMemcpyHostToDevice));

    predictIndiciesOpenGLKernel<<<numblocks, blocksize>>>(cx, cy, fx, fy, rows, cols, maxDepth, tinv, model_buffer, time, timeDelta, vmap_pi, ct_pi, nmap_pi, index_pi);

}
__global__ void predictIndiciesKernel(int* pic, int count, float cx, float cy, float fx, float fy,  int rows, int cols, float maxDepth, const mat33 Rmat_inv, const float3 tvec_inv, float* model_buffer, int time, int timeDelta, PtrStepSz<float> vmap_pi, PtrStepSz<float> ct_pi, PtrStepSz<float> nmap_pi, PtrStepSz<unsigned int> index_pi)
{

    int i = threadIdx.x + blockIdx.x * blockDim.x;

    int rows_mb, cols_mb;
    rows_mb = cols_mb = 3072;
    // int i = y* rows + x;
    float xu = 0;
    float yv = 0;

    if (i >=count)
        return;

    if (i >= rows_mb*cols_mb)
        return;


    int vz = model_buffer[i + 2*rows_mb*cols_mb];
    int cw = model_buffer[i+7*rows_mb*cols_mb];
    int vertexId;
    float3 vsrc = make_float3(__int_as_float(0x7fffffff), __int_as_float(0x7fffffff), __int_as_float(0x7fffffff));

    if (/*(vz < 0 ) ||*/ (vz > maxDepth) /*|| (time - cw > timeDelta)*/)
    {
        vsrc.x = 0;
        vsrc.y = 0;

    }
    else
    {
        float3 v_ = make_float3(__int_as_float(0x7fffffff), __int_as_float(0x7fffffff), __int_as_float(0x7fffffff));

        float3 nsrc = make_float3(__int_as_float(0x7fffffff), __int_as_float(0x7fffffff), __int_as_float(0x7fffffff));
        float3 n_ = make_float3(__int_as_float(0x7fffffff), __int_as_float(0x7fffffff), __int_as_float(0x7fffffff));

        vsrc.x = model_buffer[i];
        vsrc.y = model_buffer[i + rows_mb*cols_mb];
        vsrc.z = model_buffer[i + 2*rows_mb*cols_mb];

        nsrc.x = model_buffer[i+8*rows_mb*cols_mb];
        nsrc.y = model_buffer[i+9*rows_mb*cols_mb];
        nsrc.z = model_buffer[i+10*rows_mb*cols_mb];

        v_ = Rmat_inv * vsrc + tvec_inv;
        n_ = Rmat_inv * nsrc;
        n_ = normalized(n_);

        float3 fc;
        fc = projectPoint(v_, rows, cols, cx, cy, fx, fy, maxDepth);
        fc.x = fc.x * 0.5f + 0.5f; 
        fc.y = fc.y * 0.5f + 0.5f; 
        fc.x = fc.x * cols;
        fc.y = fc.y * rows;
        int x = fc.x, y = fc.y;

        if (x < 0 || x > cols || y < 0 || y > rows)
            return;

        atomicAdd(pic, 1);

        // printf("x = %d y = %d\n", x, y);
        vmap_pi.ptr(y)[x] = v_.x ;
        vmap_pi.ptr(y + rows)[x] = v_.y;
        vmap_pi.ptr(y + rows * 2)[x] = v_.z ;
        vmap_pi.ptr(y + rows * 3)[x] = model_buffer[i + 3*rows_mb*cols_mb];

        ct_pi.ptr(y)[x] = model_buffer[i+4*rows_mb*cols_mb];
        ct_pi.ptr(y + rows)[x] = model_buffer[i+5*rows_mb*cols_mb];
        ct_pi.ptr(y + rows * 2)[x] = model_buffer[i+6*rows_mb*cols_mb];
        ct_pi.ptr(y + rows * 3)[x] = model_buffer[i+7*rows_mb*cols_mb];

        nmap_pi.ptr(y)[x] = n_.x;
        nmap_pi.ptr(y + rows)[x] = n_.y;
        nmap_pi.ptr(y + rows * 2)[x] = n_.z;
        nmap_pi.ptr(y + rows * 3)[x] = model_buffer[i + 11*rows_mb*cols_mb];

        index_pi.ptr(y)[x] = i;
    }
}

void predictIndicies(int* pc, const CameraModel& intr, int rows, int cols, float maxDepth, const mat33& Rmat_inv, const float3& tvec_inv, DeviceArray<float>& model_buffer, int time, DeviceArray2D<float>& vmap_pi, DeviceArray2D<float>& ct_pi, DeviceArray2D<float>& nmap_pi, DeviceArray2D<unsigned int>& index_pi, int count)
{
    int blocksize = 32*8;
    int numblocks = (count + blocksize - 1)/ blocksize;
    int timeDelta = 200;

    float fx = intr.fx, cx = intr.cx;
    float fy = intr.fy, cy = intr.cy;

    float* vertices_pi;
    vertices_pi = new float[rows*cols*4];
    memset(&vertices_pi[0], 0, rows*cols*4);

    vmap_pi.create(rows*4, cols); 
    vmap_pi.upload(&vertices_pi[0], sizeof(float)*cols, 4*rows, cols);

    ct_pi.create(rows*4, cols);
    ct_pi.upload(&vertices_pi[0], sizeof(float)*cols, 4*rows, cols);

    nmap_pi.create(rows*4, cols);
    nmap_pi.upload(&vertices_pi[0], sizeof(float)*cols, 4*rows, cols);

    index_pi.create(rows,cols);
    index_pi.upload(&vertices_pi[0], sizeof(float)*cols, rows, cols);
    
    int *d_pc;
    hipMalloc((void**)&d_pc, sizeof(int));
    hipMemcpy(d_pc, pc, sizeof(int), hipMemcpyHostToDevice);
    predictIndiciesKernel<<<numblocks, blocksize>>>(d_pc, count, cx, cy, fx, fy, rows, cols, maxDepth, Rmat_inv, tvec_inv, model_buffer, time, timeDelta, vmap_pi, ct_pi, nmap_pi, index_pi);
    delete[] vertices_pi;
    cudaSafeCall(hipDeviceSynchronize());
    hipMemcpy(pc, d_pc, sizeof(int), hipMemcpyDeviceToHost);

}

__device__ bool checkNeighbours(const PtrStepSz<float> depth, int u ,int v)
{

    float z = depth.ptr(v)[u + 1];
    if(z == 0)
        return false;

    z = depth.ptr(v)[u - 1];
    if(z == 0)
        return false;

    z = depth.ptr(v - 1)[u];
    if(z == 0)
        return false;

    z = depth.ptr(v + 1)[u];
    if(z == 0)
        return false;

    return true;
}

__device__ float angleBetween(float3 a, float3 b)
{
    return acos(dot(a, b) / (sqrt(pow(a.x,2)+pow(a.y,2)+pow(a.z,2)) * sqrt(pow(b.x,2)+pow(b.y,2)+pow(b.z,2))));
}

__global__ void fusedataKernel(int* up, int* usp, const PtrStepSz<float> depth, const float* rgb, const PtrStepSz<float> depthf, float cx, float cy, float fx, float fy, int rows, int cols, float maxDepth, const mat33 Rmat, const float3 tvec, float* model_buffer, int time, PtrStepSz<float> vmap_pi, PtrStepSz<float> ct_pi, PtrStepSz<float> nmap_pi, PtrStepSz<unsigned int> index_pi, float weighting, PtrStepSz<float> updateVConf, PtrStepSz<float> updateNormRad, PtrStepSz<float> updateColTime, PtrStepSz<float> unstable_buffer)
{

    int u = threadIdx.x + blockIdx.x * blockDim.x;
    int v = threadIdx.y + blockIdx.y * blockDim.y;

    int i = v*cols + u;

    int rows_mb, cols_mb;
    rows_mb = cols_mb = 3072;
    int vCw;

    if(u < depth.cols && u > 0 && v < depth.rows && v > 0)
    {
        float z = depth.ptr(v)[u] /*/ 1000.f*/; // load and convert: mm -> meters
        if(z != 0 && z < maxDepth /*&& m == maskID*/) //FIXME
        {   
            float zf = depthf.ptr(v)[u];
            if (zf !=0 && zf < maxDepth)
            {   
                float3 vPosition_f = make_float3(zf * (u - cx) / fx,  zf * (v - cy) / fy,  zf); //depth filtering look up
                float3 cnew = make_float3(0,0,0);
                cnew.x = rgb[i];
                cnew.y = rgb[i + rows*cols];
                cnew.z = rgb[i + 2*rows*cols];
                //possibly unecessary TO DO
                if (isnan(cnew.x))
                    cnew.x = 0;
                if (isnan(cnew.y))
                    cnew.y = 0;
                if (isnan(cnew.z))
                    cnew.z = 0;

                float ec_new ;
                //FIX encoding
                cnew.x = cnew.x/255;
                cnew.y = cnew.y/255;
                cnew.z = cnew.z/255;
                ec_new = encodeColor(cnew);

                //vPosLocal = vsrc_new
                float3 vPosLocal = make_float3(z * (u - cx) / fx,  z * (v - cy) / fy,  z);
                float3 vPos_ = make_float3(0,0,0);
                vPos_ = Rmat * vPosLocal + tvec;
                float4 vPosition = make_float4(vPos_.x, vPos_.y, vPos_.z, 0); //vPosition = vnew_
                vPosition.w = confidence(cx, cy, u, v, weighting);

                float3 vNormLocal = make_float3(0,0,0);
                float3 vNorm_ = make_float3(0,0,0);
                vNormLocal = getNormal(depth, maxDepth, vPosition_f, cx, cy ,fx ,fy, u, v, rows, cols); // TO change vsrc_new  to vsrc_new_f
                vNorm_ = Rmat * vNormLocal;
                float4 vNormRad = make_float4(vNorm_.x,vNorm_.y,vNorm_.z,0);
                vNormRad.w = getRadius(fx, fy, vPosition_f.z, vNormLocal.z); // TO DO change vsrc_new.z to vsrc_new_f.z

                vCw = 0;
                int updateId = 0;
                unsigned int best = 0U;

                if(/*(int(u) % 2 == int(time) % 2) && (int(v) % 2 == int(time) % 2) && */checkNeighbours(depth, u, v) && vPosLocal.z > 0 && vPosLocal.z <= maxDepth)
                {
                    int operation = 0;
                    float bestDist = 1000;
                    float xl = (u - cx) * 1/fx;
                    float yl = (v - cy) * 1/fy;
                    float lambda = sqrt(xl * xl + yl * yl + 1);
                    float3 ray = make_float3(xl, yl, 1);

                    for (int ui = u - 2; ui < u + 2; ui++)
                    {
                        for (int vj = v - 2; vj < v + 2; vj++)
                        {
                            if ((ui < 0) || (ui >=cols))
                                continue;
                            if ((vj < 0) || (vj >=rows))
                                continue;
                            unsigned int current = index_pi.ptr(vj)[ui];
                            if(current > 0U)
                            {
                                float4 vertConf = make_float4(0,0,0,0);
                                vertConf.x = vmap_pi.ptr(vj)[ui];
                                vertConf.y = vmap_pi.ptr(vj + rows)[ui];
                                vertConf.z = vmap_pi.ptr(vj + rows * 2)[ui];
                                vertConf.w = vmap_pi.ptr(vj + rows * 3)[ui];
                                float zdiff = vertConf.z - vPosLocal.z;
                                if (abs(zdiff * lambda) < 0.05)
                                {
                                    float3 ray_v_cross = make_float3(0,0,0);
                                    ray_v_cross = cross(ray, make_float3(vertConf.x,vertConf.y,vertConf.z));
                                    float dist = sqrt(pow(ray_v_cross.x,2) + pow(ray_v_cross.y,2) + pow(ray_v_cross.z,2)) /*/ lambda*/;

                                    float4 normRad = make_float4(0,0,0,0);
                                    normRad.x = nmap_pi.ptr(vj)[ui];
                                    normRad.y = nmap_pi.ptr(vj + rows)[ui];
                                    normRad.z = nmap_pi.ptr(vj + rows * 2)[ui];
                                    normRad.w = nmap_pi.ptr(vj + rows * 3)[ui];

                                    float abw = angleBetween(make_float3(normRad.x, normRad.y, normRad.z), make_float3(vNormLocal.x, vNormLocal.y, vNormLocal.z));
                                    if(dist < bestDist && (abs(normRad.z) < 0.75f || abw < 0.5f))
                                    {
                                            operation = 1;
                                            bestDist = dist;
                                            best = current;
                                    }
                                }
                            }
                        }
                    }
                    if (operation == 1)
                    {
                        vCw = -1;
                        int intY = best / cols_mb;
                        int intX = best - (intY * cols_mb);
                        updateVConf.ptr(intY)[intX] = vPosition.x;
                        updateVConf.ptr(intY + rows_mb)[intX] = vPosition.y;
                        updateVConf.ptr(intY + rows_mb * 2)[intX] = vPosition.z;
                        updateVConf.ptr(intY + rows_mb * 3)[intX] = vPosition.w;

                        updateNormRad.ptr(intY)[intX] = vNormRad.x;
                        updateNormRad.ptr(intY + rows_mb)[intX] = vNormRad.y;
                        updateNormRad.ptr(intY + rows_mb * 2)[intX] = vNormRad.z;
                        updateNormRad.ptr(intY + rows_mb * 3)[intX] = vNormRad.w;

                        updateColTime.ptr(intY)[intX] = ec_new;
                        updateColTime.ptr(intY + rows_mb)[intX] = 0;
                        updateColTime.ptr(intY + rows_mb * 2)[intX] = time;
                        updateColTime.ptr(intY + rows_mb * 3)[intX] = vCw;

                        // unstable_buffer.ptr(v)[u] = vPosition.x;
                        // unstable_buffer.ptr(v + rows)[u] = vPosition.y;
                        // unstable_buffer.ptr(v + rows * 2)[u] = vPosition.z;
                        // unstable_buffer.ptr(v + rows * 3)[u] = vPosition.w;

                        // unstable_buffer.ptr(v + rows * 4)[u] = vNormRad.x;
                        // unstable_buffer.ptr(v + rows * 5)[u] = vNormRad.y;
                        // unstable_buffer.ptr(v + rows * 6)[u] = vNormRad.z;
                        // unstable_buffer.ptr(v + rows * 7)[u] = vNormRad.w;

                        // unstable_buffer.ptr(v + rows * 8)[u] = ec_new;
                        // unstable_buffer.ptr(v + rows * 9)[u] = 0;
                        // unstable_buffer.ptr(v + rows * 10)[u] = time;
                        // unstable_buffer.ptr(v + rows * 11)[u] = vCw;
                        atomicAdd(up, 1);
                    }
                    else
                    {
                        vCw = -2;
                        unstable_buffer.ptr(v)[u] = vPosition.x;
                        unstable_buffer.ptr(v + rows)[u] = vPosition.y;
                        unstable_buffer.ptr(v + rows * 2)[u] = vPosition.z;
                        unstable_buffer.ptr(v + rows * 3)[u] = vPosition.w;       

                        unstable_buffer.ptr(v + rows * 4)[u] = vNormRad.x;
                        unstable_buffer.ptr(v + rows * 5)[u] = vNormRad.y;
                        unstable_buffer.ptr(v + rows * 6)[u] = vNormRad.z;
                        unstable_buffer.ptr(v + rows * 7)[u] = vNormRad.w;

                        unstable_buffer.ptr(v + rows * 8)[u] = ec_new;
                        unstable_buffer.ptr(v + rows * 9)[u] = 0;
                        unstable_buffer.ptr(v + rows * 10)[u] = time;
                        unstable_buffer.ptr(v + rows * 11)[u] = vCw;     
                        atomicAdd(usp, 1);

                    }
                }
            }
        }
    }
}

void fuse_data(int* up, int* usp, DeviceArray2D<float>& depth,  DeviceArray<float>& rgb, DeviceArray2D<float>& depthf, const CameraModel& intr, int rows, int cols, float maxDepth, const mat33& Rmat, const float3& tvec, DeviceArray<float>& model_buffer, int time, DeviceArray2D<float>& vmap_pi, DeviceArray2D<float>& ct_pi, DeviceArray2D<float>& nmap_pi, DeviceArray2D<unsigned int>& index_pi, float weighting, DeviceArray2D<float>& updateVConf, DeviceArray2D<float>& updateNormRad, DeviceArray2D<float>& updateColTime, DeviceArray2D<float>& unstable_buffer)
{
    dim3 block (32, 8);
    dim3 grid (1, 1, 1);
    grid.x = getGridDim (depth.cols (), block.x);
    grid.y = getGridDim (depth.rows (), block.y);

    float fx = intr.fx, cx = intr.cx;
    float fy = intr.fy, cy = intr.cy;

    int *d_up, *d_usp;
    hipMalloc((void**)&d_up, sizeof(int));
    hipMalloc((void**)&d_usp, sizeof(int));
    hipMemcpy(d_up, up, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_usp, usp, sizeof(int), hipMemcpyHostToDevice);

    fusedataKernel<<<grid, block>>>(d_up, d_usp, depth, rgb, depthf, cx, cy, fx, fy, rows, cols, maxDepth, Rmat, tvec, model_buffer, time, vmap_pi, ct_pi, nmap_pi, index_pi, weighting, updateVConf, updateNormRad, updateColTime, unstable_buffer);
    cudaSafeCall(hipGetLastError());
    cudaSafeCall(hipDeviceSynchronize());
    hipMemcpy(up, d_up, sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(usp, d_usp, sizeof(int), hipMemcpyDeviceToHost);


}


__global__ void fuseupdateKernel(int* cvw0, int* cvwm1, float cx, float cy, float fx, float fy, int rows, int cols, float maxDepth, const mat33 Rmat, const float3 tvec , float* model_buffer, float* model_buffer_rs, int time, PtrStepSz<float> updateVConf, PtrStepSz<float> updateNormRad, PtrStepSz<float> updateColTime)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int rows_mb, cols_mb;
    rows_mb = cols_mb = 3072;

    if (i >= rows_mb*cols_mb)
        return;

    int intY = i / cols_mb;
    int intX = i - (intY * cols_mb);

    float newColx = updateColTime.ptr(intY)[intX];
    float cVw =  updateColTime.ptr(intY + 3*rows_mb)[intX];

    if (cVw == 0)
    {
        atomicAdd(cvw0, 1);
        model_buffer_rs[i] = model_buffer[i];
        model_buffer_rs[i+ rows_mb*cols_mb] = model_buffer[i+ rows_mb*cols_mb];
        model_buffer_rs[i+2*rows_mb*cols_mb] = model_buffer[i+2*rows_mb*cols_mb];
        model_buffer_rs[i+3*rows_mb*cols_mb] = model_buffer[i+3*rows_mb*cols_mb];

        // //writing color and time
        model_buffer_rs[i+4*rows_mb*cols_mb] =  model_buffer[i+4*rows_mb*cols_mb];//x
        model_buffer_rs[i+5*rows_mb*cols_mb] =  model_buffer[i+5*rows_mb*cols_mb];//y
        model_buffer_rs[i+6*rows_mb*cols_mb] =  model_buffer[i+6*rows_mb*cols_mb];//z
        model_buffer_rs[i+7*rows_mb*cols_mb] =  model_buffer[i+7*rows_mb*cols_mb];

        //writing normals
        model_buffer_rs[i+8*rows_mb*cols_mb] = model_buffer[i+8*rows_mb*cols_mb];
        model_buffer_rs[i+9*rows_mb*cols_mb] = model_buffer[i+9*rows_mb*cols_mb];
        model_buffer_rs[i+10*rows_mb*cols_mb] = model_buffer[i+10*rows_mb*cols_mb];
        model_buffer_rs[i+11*rows_mb*cols_mb] = model_buffer[i+11*rows_mb*cols_mb];

    }
    else if (cVw == -1)
    {
        atomicAdd(cvwm1, 1);

        float4 newNorm = make_float4(updateNormRad.ptr(intY)[intX], updateNormRad.ptr(intY + rows_mb)[intX], updateNormRad.ptr(intY + 2*rows_mb)[intX], updateNormRad.ptr(intY + 3*rows_mb)[intX]);
        float4 vNormRad = make_float4(model_buffer[i+8*rows_mb*cols_mb], model_buffer[i+9*rows_mb*cols_mb], model_buffer[i+10*rows_mb*cols_mb], model_buffer[i+11*rows_mb*cols_mb]);

        float a = updateVConf.ptr(intY+3*rows_mb)[intX];
        float3 v_g = make_float3(updateVConf.ptr(intY)[intX], updateVConf.ptr(intY + rows_mb)[intX], updateVConf.ptr(intY + 2*rows_mb)[intX]);
        float c_k = model_buffer[i+3*rows_mb*cols_mb];
        float3 v_k = make_float3(model_buffer[i], model_buffer[i+ rows_mb*cols_mb], model_buffer[i+ 2*rows_mb*cols_mb]);
        if (newNorm.w < (1 + 0.5) * vNormRad.w)
        {
            model_buffer_rs[i] = (c_k * v_k.x + a * v_g.x) / (c_k + a);
            model_buffer_rs[i+ rows_mb*cols_mb] = (c_k * v_k.y + a * v_g.y) / (c_k + a);
            model_buffer_rs[i+2*rows_mb*cols_mb] = (c_k * v_k.z + a * v_g.z) / (c_k + a);
            model_buffer_rs[i+3*rows_mb*cols_mb] = c_k + a;

            // TO DO color add
            float3 oldCol = decodeColor(model_buffer[i+4*rows_mb*cols_mb]);
            float3 newCol = decodeColor(newColx);
            
            float3 avgColor = make_float3((c_k * oldCol.x+ a * newCol.x)/ (c_k + a), (c_k * oldCol.y+ a * newCol.y)/ (c_k + a), (c_k * oldCol.z+ a * newCol.z)/ (c_k + a));
            float4 vColor0 = make_float4(encodeColor(avgColor), model_buffer_rs[i+5*rows_mb*cols_mb], model_buffer_rs[i+6*rows_mb*cols_mb], time);

            model_buffer_rs[i+4*rows_mb*cols_mb] = vColor0.x; 
            model_buffer_rs[i+5*rows_mb*cols_mb] = vColor0.y;
            model_buffer_rs[i+6*rows_mb*cols_mb] = vColor0.z;
            model_buffer_rs[i+7*rows_mb*cols_mb] = vColor0.w;

            float4 vNormRad0 = make_float4((c_k * vNormRad.x+ a * newNorm.x)/ (c_k + a), (c_k * vNormRad.y+ a * newNorm.y)/ (c_k + a), (c_k * vNormRad.z+ a * newNorm.z)/ (c_k + a), (c_k * vNormRad.w+ a * newNorm.w)/ (c_k + a));
            float3 normnrad = normalized(make_float3(vNormRad0.x,vNormRad0.y,vNormRad0.z));
            model_buffer_rs[i+8*rows_mb*cols_mb] = normnrad.x;
            model_buffer_rs[i+9*rows_mb*cols_mb] = normnrad.y;
            model_buffer_rs[i+10*rows_mb*cols_mb] = normnrad.z;
            model_buffer_rs[i+11*rows_mb*cols_mb] = vNormRad0.w;

        }
        else
        {
            model_buffer_rs[i] = model_buffer[i];
            model_buffer_rs[i+ rows_mb*cols_mb] = model_buffer[i+ rows_mb*cols_mb];
            model_buffer_rs[i+2*rows_mb*cols_mb] = model_buffer[i+2*rows_mb*cols_mb];
            model_buffer_rs[i+3*rows_mb*cols_mb] = model_buffer[i+3*rows_mb*cols_mb];

            // //writing color and time
            model_buffer_rs[i+4*rows_mb*cols_mb] =  model_buffer[i+4*rows_mb*cols_mb]; //x
            model_buffer_rs[i+5*rows_mb*cols_mb] =  model_buffer[i+5*rows_mb*cols_mb];//y
            model_buffer_rs[i+6*rows_mb*cols_mb] =  model_buffer_rs[i+6*rows_mb*cols_mb];//z
            model_buffer_rs[i+7*rows_mb*cols_mb] =  model_buffer[i+7*rows_mb*cols_mb];

            //writing normals
            model_buffer_rs[i+8*rows_mb*cols_mb] = model_buffer[i+8*rows_mb*cols_mb];
            model_buffer_rs[i+9*rows_mb*cols_mb] = model_buffer[i+9*rows_mb*cols_mb];
            model_buffer_rs[i+10*rows_mb*cols_mb] = model_buffer[i+10*rows_mb*cols_mb];
            model_buffer_rs[i+11*rows_mb*cols_mb] = model_buffer[i+11*rows_mb*cols_mb];

            model_buffer_rs[i+3*rows_mb*cols_mb] = c_k + a;
            model_buffer_rs[i+7*rows_mb*cols_mb]= time;

        }
    }
}

void fuse_update(int* cvw0, int* cvwm1, const CameraModel& intr, int rows, int cols, float maxDepth, const mat33& Rmat, const float3& tvec, DeviceArray<float>& model_buffer, DeviceArray<float>& model_buffer_rs, int time, int* h_count, DeviceArray2D<float>& updateVConf, DeviceArray2D<float>& updateNormRad, DeviceArray2D<float>& updateColTime)
{

    int blocksize = 32*8;
    int numblocks = (*h_count + blocksize - 1)/ blocksize;
    float fx = intr.fx, fy = intr.fy, cx = intr.cx, cy = intr.cy;

    int *d_cvw0, *d_cvwm1;
    hipMalloc((void**)&d_cvw0, sizeof(int));
    hipMalloc((void**)&d_cvwm1, sizeof(int));
    hipMemcpy(d_cvw0, cvw0, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_cvwm1, cvwm1, sizeof(int), hipMemcpyHostToDevice);

    fuseupdateKernel<<<numblocks, blocksize>>>(d_cvw0, d_cvwm1, cx, cy, fx, fy, rows, cols, maxDepth, Rmat, tvec, model_buffer, model_buffer_rs, time, updateVConf, updateNormRad, updateColTime);
    cudaSafeCall(hipGetLastError());
    hipMemcpy(cvw0, d_cvw0, sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(cvwm1, d_cvwm1, sizeof(int), hipMemcpyDeviceToHost);

}

__global__ void cleanKernel2D(const PtrStepSz<float> depthf, float cx, float cy, float fx, float fy, int rows, int cols, float maxDepth, const mat33 Rmat_inv, const float3 tvec_inv, float* model_buffer, int* d_count, int fixed_count, int time, int timeDelta, float confThreshold, PtrStepSz<float> vmap_pi, PtrStepSz<float> ct_pi, PtrStepSz<float> nmap_pi, PtrStepSz<unsigned int> index_pi, PtrStepSz<float> updateVConf, PtrStepSz<float> updateNormRad, PtrStepSz<float> updateColTime, PtrStepSz<float> unstable_buffer)
{   

    int u = threadIdx.x + blockIdx.x * blockDim.x;
    int v = threadIdx.y + blockIdx.y * blockDim.y;

    int i = v*cols + u;

    int rows_mb, cols_mb;
    rows_mb = cols_mb = 3072;

    int test = 1;

    if(fixed_count + i >= rows_mb*cols_mb)
        return;

    if(u < cols && u > 0 && v < rows && v > 0)
    {
        float4 vPosition, vNormRad, vColor, vertConf, colorTime, normRad;
        float3 localPos, localNorm, vPosition_xyz, vNormRad_xyz;
        vertConf = make_float4(0,0,0,0);
        colorTime = make_float4(0,0,0,0);
        normRad = make_float4(0,0,0,0);

        vPosition = make_float4(unstable_buffer.ptr(v)[u], unstable_buffer.ptr(v + rows)[u], unstable_buffer.ptr(v + 2*rows)[u], unstable_buffer.ptr(v + 3*rows)[u]);
        vNormRad = make_float4(unstable_buffer.ptr(v + 4*rows)[u], unstable_buffer.ptr(v + 5*rows)[u], unstable_buffer.ptr(v + 6*rows)[u], unstable_buffer.ptr(v + 7*rows)[u]);
        vColor = make_float4(unstable_buffer.ptr(v + 8*rows)[u], unstable_buffer.ptr(v + 9*rows)[u], unstable_buffer.ptr(v + 10*rows)[u], unstable_buffer.ptr(v + 11*rows)[u]);
    
        if(vPosition.x == 0 && vPosition.y == 0 && vPosition.z == 0)
            return;

        vPosition_xyz = make_float3(vPosition.x,vPosition.y,vPosition.z);
        localPos = make_float3(0,0,0);
        localPos = Rmat_inv * vPosition_xyz + tvec_inv;
    
        float x = ((fx * localPos.x) / localPos.z) + cx;
        float y = ((fy * localPos.y) / localPos.z) + cy;

        if (x < 0 || x > cols || y < 0 || y > rows)
            return;
    
        localNorm = make_float3(0,0,0);
        vNormRad_xyz = make_float3(vNormRad.x, vNormRad.y, vNormRad.z);
        localNorm = Rmat_inv * vNormRad_xyz;
 
        int count = 0;
        int zCount = 0;
        int violationCount = 0; // Look-through outlier test
        float avgViolation = 0;
        float outlierCoeff = 0.9;

        if(/*(time - vColor.w < timeDelta) &&*/ (localPos.z > 0) && (x > 0) && (y > 0) && (x < cols) && (y < rows))
        {
            for(int ui = x - 2; ui < x + 2; ui++){
                for(int vj = y - 2; vj < y + 2; vj++){

                  unsigned int current = index_pi.ptr(vj)[ui];
                   if(current > 0U)
                   {
                        vertConf.x = vmap_pi.ptr(vj)[ui];
                        vertConf.y = vmap_pi.ptr(vj + rows)[ui];
                        vertConf.z = vmap_pi.ptr(vj + rows * 2)[ui];
                        vertConf.w = vmap_pi.ptr(vj + rows * 3)[ui];

                        colorTime.x = ct_pi.ptr(vj)[ui];
                        colorTime.y = ct_pi.ptr(vj + rows)[ui];
                        colorTime.z = ct_pi.ptr(vj + rows * 2)[ui];
                        colorTime.w = ct_pi.ptr(vj + rows * 3)[ui];

                        normRad.x = nmap_pi.ptr(vj)[ui];
                        normRad.y = nmap_pi.ptr(vj + rows)[ui];
                        normRad.z = nmap_pi.ptr(vj + rows * 2)[ui];
                        normRad.w = nmap_pi.ptr(vj + rows * 3)[ui];

                        float cond = sqrt(dot(make_float3(vertConf.x-localPos.x, vertConf.y-localPos.y, 0), make_float3(vertConf.x-localPos.x, vertConf.y-localPos.y, 0)));

                       if(/*(colorTime.z < vColor.z) && */// Surfel in map is older (init-time)
                          (vertConf.w > confThreshold) && // Surfel in map is good (high conf)
                          (vertConf.z > localPos.z) && // Surfel in map is behind vertex
                          (vertConf.z - localPos.z < 0.01) && // Close to each other
                          (cond < vNormRad.w * 1.4)){ // falls within radius
                           count++;
                       }
                       
                       if(/*(colorTime.w == time) && */// Only possible if lost?
                          (vertConf.w > confThreshold) && // Surfel in map is good (high conf)
                          (vertConf.z > localPos.z) && // Surfel in map is behind vertex
                          (vertConf.z - localPos.z > 0.01) && // Not too close
                          (abs(localNorm.z) > 0.85f)){
                           zCount++;
                       }
                   }
                }
            }

            // New outlier rejection ("see-through")
            for(int ui = x - 2; ui <= x + 2; ui++){
                for(int vj = y - 2; vj <= y + 2; vj++){
                    float d = depthf.ptr(vj)[ui] - localPos.z; //cast depthf float? TO DO
                    if(d > 0.03) {
                      violationCount++;
                      avgViolation += d;
                    }
                }
            }
        }
        
        if((count > 8) || (zCount > 4)) test = 0;
        
        //New unstable point
        if(vColor.w == -2) vColor.w = time;
        
        //Degenerate case or too unstable
        if(((vColor.w == -1) || (((time - vColor.w) > 20) && (vPosition.w < confThreshold)))) test = 0;
        
        if((vColor.w > 0 ) && (time - vColor.w) > timeDelta) test = 1;

        if(violationCount > 0) 
        {
          avgViolation /= violationCount;
          vPosition.w *= 1.0 / (1 + outlierCoeff * avgViolation);
          //uint maskValue = uint(textureLod(maskSampler, vec2(x_n, y_n), 0.0));
          float wDepth = float(depthf.ptr(int(y))[int(x)]);
          if(wDepth > 0.0f && wDepth > localPos.z+0.03) violationCount++;
          if((wDepth > localPos.z-0.05 && wDepth < localPos.z+0.05)) vPosition.w *= (0.5 + 0.5 * (1 - outlierCoeff / 10.0));
        }

        if (test == 1)
        {
            model_buffer[fixed_count+i] = vPosition.x;
            model_buffer[fixed_count+i+rows_mb*cols_mb] = vPosition.y;
            model_buffer[fixed_count+i+2*rows_mb*cols_mb] = vPosition.z;
            model_buffer[fixed_count+i+3*rows_mb*cols_mb] = vPosition.w;

            model_buffer[fixed_count+i+4*rows_mb*cols_mb] = vColor.x;
            model_buffer[fixed_count+i+5*rows_mb*cols_mb] = vColor.y;
            model_buffer[fixed_count+i+6*rows_mb*cols_mb] = vColor.z;
            model_buffer[fixed_count+i+7*rows_mb*cols_mb] = vColor.w;

            model_buffer[fixed_count+i+8*rows_mb*cols_mb] = vNormRad.x;
            model_buffer[fixed_count+i+9*rows_mb*cols_mb] = vNormRad.y;
            model_buffer[fixed_count+i+10*rows_mb*cols_mb] = vNormRad.z;
            model_buffer[fixed_count+i+11*rows_mb*cols_mb] = vNormRad.w;


            // printf("vx = %f vy = %f vz = %f vw = %f cx = %f cy = %f cz = %f cw = %f nx = %f ny = %f nz = %f nw = %f\n",model_buffer[*d_count],model_buffer[*d_count+ rows_mb*cols_mb], model_buffer[*d_count+ 2*rows_mb*cols_mb], model_buffer[*d_count+ 3*rows_mb*cols_mb], model_buffer[*d_count+4*rows_mb*cols_mb], model_buffer[*d_count+5*rows_mb*cols_mb], model_buffer[*d_count+6*rows_mb*cols_mb], model_buffer[*d_count+ 7*rows_mb*cols_mb], model_buffer[*d_count+8*rows_mb*cols_mb], model_buffer[*d_count+9*rows_mb*cols_mb], model_buffer[*d_count+10*rows_mb*cols_mb], model_buffer[*d_count+11*rows_mb*cols_mb]);
            atomicAdd(d_count, 1);

        }
    }
}   
__global__ void cleanKernel1D(const PtrStepSz<float> depthf, float cx, float cy, float fx, float fy, int rows, int cols, float maxDepth, const mat33 Rmat_inv, const float3 tvec_inv, float* model_buffer, float* model_buffer_rs, int time, int timeDelta, float confThreshold, PtrStepSz<float> vmap_pi, PtrStepSz<float> ct_pi, PtrStepSz<float> nmap_pi, PtrStepSz<unsigned int> index_pi, PtrStepSz<float> updateVConf, PtrStepSz<float> updateNormRad, PtrStepSz<float> updateColTime)
{   

    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int rows_mb, cols_mb;
    rows_mb = cols_mb = 3072;

    int test = 1;

    if((i > 0) && (i < rows_mb*cols_mb))
    {
        float4 vPosition, vNormRad, vColor, vertConf, colorTime, normRad;
        float3 localPos, localNorm, vPosition_xyz, vNormRad_xyz; 
        vertConf = make_float4(0,0,0,0);
        colorTime = make_float4(0,0,0,0);
        normRad = make_float4(0,0,0,0);

        vPosition = make_float4(model_buffer_rs[i], model_buffer_rs[i+ rows_mb*cols_mb], model_buffer_rs[i+2*rows_mb*cols_mb], model_buffer_rs[i+3*rows_mb*cols_mb]);
        vNormRad = make_float4(model_buffer_rs[i+8*rows_mb*cols_mb], model_buffer_rs[i+9*rows_mb*cols_mb], model_buffer_rs[i+10*rows_mb*cols_mb], model_buffer_rs[i+11*rows_mb*cols_mb]);
        vColor = make_float4(model_buffer_rs[i+4*rows_mb*cols_mb], model_buffer_rs[i+5*rows_mb*cols_mb], model_buffer_rs[i+6*rows_mb*cols_mb], model_buffer_rs[i+7*rows_mb*cols_mb]);
        
        vPosition_xyz = make_float3(vPosition.x,vPosition.y,vPosition.z);
        localPos = make_float3(0,0,0);
        localPos = Rmat_inv * vPosition_xyz + tvec_inv;
    
        float x = ((fx * localPos.x) / localPos.z) + cx;
        float y = ((fy * localPos.y) / localPos.z) + cy;

        if (x < 0 || x > cols || y < 0 || y > rows)
            return;
    
        localNorm = make_float3(0,0,0);
        vNormRad_xyz = make_float3(vNormRad.x, vNormRad.y, vNormRad.z);
        localNorm = Rmat_inv * vNormRad_xyz;
 
        int count = 0;
        int zCount = 0;
        int violationCount = 0; // Look-through outlier test
        float avgViolation = 0;
        float outlierCoeff = 0.9;

        if(/*(time - vColor.w < timeDelta) && */(localPos.z > 0) && (x > 0) && (y > 0) && (x < cols) && (y < rows))
        {
            for(int ui = x - 2; ui < x + 2; ui++){
                for(int vj = y - 2; vj < y + 2; vj++){

                  unsigned int current = index_pi.ptr(vj)[ui];
                   if(current > 0U)
                   {
                        vertConf.x = vmap_pi.ptr(vj)[ui];
                        vertConf.y = vmap_pi.ptr(vj + rows)[ui];
                        vertConf.z = vmap_pi.ptr(vj + rows * 2)[ui];
                        vertConf.w = vmap_pi.ptr(vj + rows * 3)[ui];

                        colorTime.x = ct_pi.ptr(vj)[ui];
                        colorTime.y = ct_pi.ptr(vj + rows)[ui];
                        colorTime.z = ct_pi.ptr(vj + rows * 2)[ui];
                        colorTime.w = ct_pi.ptr(vj + rows * 3)[ui];

                        normRad.x = nmap_pi.ptr(vj)[ui];
                        normRad.y = nmap_pi.ptr(vj + rows)[ui];
                        normRad.z = nmap_pi.ptr(vj + rows * 2)[ui];
                        normRad.w = nmap_pi.ptr(vj + rows * 3)[ui];

                        float cond = sqrt(dot(make_float3(vertConf.x-localPos.x, vertConf.y-localPos.y, 0), make_float3(vertConf.x-localPos.x, vertConf.y-localPos.y, 0)));

                        if((colorTime.z < vColor.z) && // Surfel in map is older (init-time)
                          (vertConf.w > confThreshold) && // Surfel in map is good (high conf)
                          (vertConf.z > localPos.z) && // Surfel in map is behind vertex
                          (vertConf.z - localPos.z < 0.01) && // Close to each other
                          (cond < vNormRad.w * 1.4)){ // falls within radius
                           count++;
                        }

                        if((colorTime.w == time) && // Only possible if lost?
                          (vertConf.w > confThreshold) && // Surfel in map is good (high conf)
                          (vertConf.z > localPos.z) && // Surfel in map is behind vertex
                          (vertConf.z - localPos.z > 0.01) && // Not too close
                          (abs(localNorm.z) > 0.85f)){
                           zCount++;
                        }
                   }
                }
            }

            // New outlier rejection ("see-through")
            for(int ui = x - 2; ui <= x + 2; ui++){
                for(int vj = y - 2; vj <= y + 2; vj++){
                    float d = depthf.ptr(vj)[ui] - localPos.z; //cast depthf float? TO DO
                    if(d > 0.03) {
                      violationCount++;
                      avgViolation += d;
                    }
                }
            }
        }
        
        if((count > 8) || (zCount > 4)) test = 0;
        
        //New unstable point
        if(vColor.w == -2) vColor.w = time;
        
        //Degenerate case or too unstable
        if(((vColor.w == -1) || (((time - vColor.w) > 20) && (vPosition.w < confThreshold)))) test = 0;
        
        if((vColor.w > 0 ) && (time - vColor.w) > timeDelta) test = 1;

        if(violationCount > 0) 
        {
          avgViolation /= violationCount;
          vPosition.w *= 1.0 / (1 + outlierCoeff * avgViolation);
          //uint maskValue = uint(textureLod(maskSampler, vec2(x_n, y_n), 0.0));
          float wDepth = float(depthf.ptr(int(y))[int(x)]);
          if(wDepth > 0.0f && wDepth > localPos.z+0.03) violationCount++;
          if((wDepth > localPos.z-0.05 && wDepth < localPos.z+0.05)) vPosition.w *= (0.5 + 0.5 * (1 - outlierCoeff / 10.0));
        }

        if (test == 1)
        {
            // write
            model_buffer[i] = vPosition.x;
            model_buffer[i+ rows_mb*cols_mb] = vPosition.y;
            model_buffer[i+2*rows_mb*cols_mb] = vPosition.z;
            model_buffer[i+3*rows_mb*cols_mb] = vPosition.w;

            model_buffer[i+8*rows_mb*cols_mb] = vNormRad.x;
            model_buffer[i+9*rows_mb*cols_mb] = vNormRad.y;
            model_buffer[i+10*rows_mb*cols_mb] = vNormRad.z;
            model_buffer[i+11*rows_mb*cols_mb] = vNormRad.w;

            model_buffer[i+4*rows_mb*cols_mb] = vColor.x;
            model_buffer[i+5*rows_mb*cols_mb] = vColor.y;
            model_buffer[i+6*rows_mb*cols_mb] = vColor.z;
            model_buffer[i+7*rows_mb*cols_mb] = vColor.w;
            // printf("vx = %f vy = %f vz = %f vw = %f cx = %f cy = %f cz = %f cw = %f nx = %f ny = %f nz = %f nw = %f\n",model_buffer[i],model_buffer[i+ rows_mb*cols_mb], model_buffer[i+ 2*rows_mb*cols_mb], model_buffer[i+ 3*rows_mb*cols_mb], model_buffer[i+4*rows_mb*cols_mb], model_buffer[i+5*rows_mb*cols_mb], model_buffer[i+6*rows_mb*cols_mb], model_buffer[i+ 7*rows_mb*cols_mb], model_buffer[i+8*rows_mb*cols_mb], model_buffer[i+9*rows_mb*cols_mb], model_buffer[i+10*rows_mb*cols_mb], model_buffer[i+11*rows_mb*cols_mb]);

        }

    }
}   

void clean(DeviceArray2D<float>& depthf, const CameraModel& intr, int rows, int cols, float maxDepth, const mat33 Rmat_inv, const float3 tvec_inv, DeviceArray<float>& model_buffer, DeviceArray<float>& model_buffer_rs, int time, int timeDelta, float confThreshold, int * h_count, DeviceArray2D<float>& vmap_pi, DeviceArray2D<float>& ct_pi, DeviceArray2D<float>& nmap_pi, DeviceArray2D<unsigned int>& index_pi, DeviceArray2D<float>& updateVConf, DeviceArray2D<float>& updateNormRad, DeviceArray2D<float>& updateColTime, DeviceArray2D<float>& unstable_buffer)
{

    int blocksize = 32*8;
    int numblocks = (*h_count + blocksize - 1)/ blocksize;
    float fx = intr.fx, fy = intr.fy, cx = intr.cx, cy = intr.cy;
    cleanKernel1D<<<numblocks, blocksize>>>(depthf, cx, cy, fx, fy, rows, cols,  maxDepth, Rmat_inv, tvec_inv, model_buffer, model_buffer_rs, time, timeDelta, confThreshold, vmap_pi, ct_pi, nmap_pi, index_pi, updateVConf, updateNormRad, updateColTime);
    cudaSafeCall(hipGetLastError());
    dim3 grid (1, 1, 1);
    grid.x = getGridDim (depthf.cols (), 32);
    grid.y = getGridDim (depthf.rows (), 8);
    //check count TO DO
    int *d_count, fixed_count;
    fixed_count = *h_count;
    hipMalloc((void**)&d_count, sizeof(int));
    hipMemcpy(d_count, h_count, sizeof(int), hipMemcpyHostToDevice);
    cleanKernel2D<<<grid, blocksize>>>(depthf, cx, cy, fx, fy, rows, cols, maxDepth, Rmat_inv, tvec_inv, model_buffer, d_count, fixed_count, time, timeDelta, confThreshold, vmap_pi, ct_pi, nmap_pi, index_pi, updateVConf, updateNormRad, updateColTime, unstable_buffer);
    cudaSafeCall(hipGetLastError());
    cudaSafeCall(hipDeviceSynchronize());
    hipMemcpy(h_count, d_count, sizeof(int), hipMemcpyDeviceToHost);
}


__global__ void testimagecopyKernel(float* rgb, float* imagebin, int cols, int rows, int ibcount)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;

    if (i > 0 && i < rows*cols)
    {   
        int x, y;
        y = i/cols;
        x = i - y*cols;
        float r = rgb[3*y*cols + 3*x];
        imagebin[i+(ibcount*rows*cols)] = r;
    }

}

void testimagecopy(DeviceArray<float> rgb, DeviceArray<float> imagebin,  int cols, int rows, int ibcount)
{   
    int blocksize = 32*8;
    int numblocks = (rows*cols)/ blocksize;

    // int* d_ibcount;
    // hipMalloc((void**)&d_ibcount, sizeof(int));
    // hipMemcpy(d_ibcount, ibcount, sizeof(int), hipMemcpyHostToDevice);
    testimagecopyKernel<<<numblocks, blocksize>>>(rgb, imagebin, cols, rows, ibcount);
    cudaSafeCall(hipGetLastError());
    // hipDeviceSynchronize();
    // hipMemcpy(ibcount, d_ibcount, sizeof(int), hipMemcpyDeviceToHost);

}

__global__ void testimageprintKernel(float* imagebin, int ibcount)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    float r = imagebin[i];
    
}

void testimageprint(DeviceArray<float> imagebin, int cols, int rows, int ibcount)
{   
    int blocksize = 32*8;
    int numblocks = ((ibcount)*rows*cols)/ blocksize;

    // int* d_ibcount;
    // hipMalloc((void**)&d_ibcount, sizeof(int));
    // hipMemcpy(d_ibcount, ibcount, sizeof(int), hipMemcpyHostToDevice);
    testimageprintKernel<<<numblocks, blocksize>>>(imagebin, ibcount);
    cudaSafeCall(hipGetLastError());
    // hipMemcpy(ibcount, d_ibcount, sizeof(int), hipMemcpyDeviceToHost);

}
__global__ void testcolorencodingKernel()
{
    float3 c = make_float3(59,74,43);
    float ec = encodeColor(c);
    float3 dc = decodeColor(ec);
    printf("org color %f %f %f\n encoded color %f decoded color %f %f %f \n ",c.x,c.y,c.z, ec, dc.x,dc.y,dc.z);
}
void testcolorencoding()
{
    testcolorencodingKernel<<<1,1>>>();
}

__global__ void normalFusionDataKernel(float* model_buffer, int* count, int fixed_count, int* update_count, int time, const PtrStepSz<float> depth, float fx, float fy, float cx, float cy, int rows, int cols, float maxDepth, const mat33 Rmat, const float3 tvec, float weighting, PtrStepSz<float> vmap_pi, PtrStepSz<float> ct_pi, PtrStepSz<float> nmap_pi, PtrStepSz<unsigned int> index_pi, PtrStepSz<float> neighbours_and_vert)
{
    
    int u = threadIdx.x + blockIdx.x * blockDim.x;
    int v = threadIdx.y + blockIdx.y * blockDim.y;

    int i = v*cols + u;

    int rows_mb, cols_mb;
    rows_mb = cols_mb = 3072;

    // if (*count >= rows_mb*cols_mb/15)
    //     return;

    if(u < depth.cols && u > 0 && v < depth.rows && v > 0)
    {
        float z = depth.ptr(v)[u] /*/ 1000.f*/; // load and convert: mm -> meters

        if(z != 0 && z < maxDepth /*&& m == maskID*/) //FIXME
        {  

            unsigned int best = 0U;
            float3 vPosLocal = make_float3(z * (u - cx) / fx,  z * (v - cy) / fy,  z);
            // printf("vPosLocal x=%f y=%f z=%f\n",vPosLocal.x, vPosLocal.y, vPosLocal.z);
            float3 vPosition_ = make_float3(0,0,0); //vPosition = vnew_
            float4 vPosition = make_float4(0,0,0,0); //vPosition = vnew_
            
            vPosition_ = Rmat * vPosLocal + tvec;
            vPosition.x = vPosition_.x;
            vPosition.y = vPosition_.y;
            vPosition.z = vPosition_.z;
            vPosition.w = confidence(cx, cy, u, v, weighting);

            float3 vNormLocal = make_float3(0,0,0);
            //nnew_ = vNormRad
            float4 vNormRad = make_float4(0,0,0,0);
            float3 vNormRad_ = make_float3(0,0,0);

            float rnew;
            vNormLocal = getNormal(depth, maxDepth, vPosLocal, cx, cy ,fx ,fy, u, v, rows, cols); // TO change vsrc_new  to vsrc_new_f
            vNormRad_ = Rmat * vNormLocal;
            vNormRad.x = vNormRad_.x;
            vNormRad.y = vNormRad_.y;
            vNormRad.z = vNormRad_.z;
            vNormRad.w = getRadius(fx, fy, vPosLocal.z, vNormLocal.z); // TO DO change vsrc_new.z to vsrc_new_f.z

            if(/*(int(u) % 2 == int(time) % 2) && (int(v) % 2 == int(time) % 2) && */checkNeighbours(depth, u, v) && vPosLocal.z > 0 && vPosLocal.z <= maxDepth)
            {
                int operation = 0;
                float bestDist = 1000;
                float xl = (u - cx) * 1/fx;
                float yl = (v - cy) * 1/fy;
                float lambda = sqrt(xl * xl + yl * yl + 1);
                float3 ray = make_float3(xl, yl, 1);
                float4 vertConf = make_float4(0,0,0,0);
                float4 normRad = make_float4(0,0,0,0);

                for (int ui = u - 2; ui < u + 2; ui++)
                {
                    for (int vj = v - 2; vj < v + 2; vj++)
                    {
                        if ((ui < 0) || (ui >=cols))
                            continue;
                        if ((vj < 0) || (vj >=rows))
                            continue;
                        unsigned int current = index_pi.ptr(vj)[ui];
                        if(current > 0U)
                        {
                            vertConf.x = vmap_pi.ptr(vj)[ui];
                            vertConf.y = vmap_pi.ptr(vj + rows)[ui];
                            vertConf.z = vmap_pi.ptr(vj + rows * 2)[ui];
                            // vertConf.w = vmap_pi.ptr(vj + rows * 3)[ui];
                             
                            float zdiff = vertConf.z - vPosLocal.z;

                            if (abs(zdiff * lambda) < 0.05)
                            {
                                float3 ray_v_cross = make_float3(0,0,0);
                                ray_v_cross = cross(ray, make_float3(vertConf.x,vertConf.y,vertConf.z));
                                float dist = sqrt(pow(ray_v_cross.x,2) + pow(ray_v_cross.y,2) + pow(ray_v_cross.z,2)) /*/ lambda*/;

                                normRad.x = nmap_pi.ptr(vj)[ui];
                                normRad.y = nmap_pi.ptr(vj + rows)[ui];
                                normRad.z = nmap_pi.ptr(vj + rows * 2)[ui];
                                normRad.w = nmap_pi.ptr(vj + rows * 3)[ui];
                                
                                float abw = angleBetween(make_float3(normRad.x, normRad.y, normRad.z), make_float3(vNormLocal.x, vNormLocal.y, vNormLocal.z));
                                
                                if(dist < bestDist && (abs(normRad.z) < 0.75f || abw < 0.5f))
                                {
                                        operation = 1;
                                        bestDist = dist;
                                        best = current;
                                }
                            }
                        }
                    }
                }
                if (operation == 1)
                {

                    float3 vConf_ = make_float3(vertConf.x, vertConf.y, vertConf.z);
                    float3 vConf = make_float3(0, 0, 0);

                    vConf = Rmat*vConf_ + tvec;
                    
                    float3 nRad_ = make_float3(normRad.x, normRad.y, normRad.z);
                    float3 nRad = make_float3(0, 0, 0);

                    nRad = Rmat * nRad_;

                    float a = vPosition.w;
                    float3 v_g = make_float3(vPosition.x, vPosition.y, vPosition.z);
                    float c_k = vertConf.w;
                    float3 v_k = make_float3(vConf.x, vConf.y, vConf.z);


                    if (vNormRad.w < (1 + .5) * normRad.w)
                    {
                        atomicAdd(update_count, 1);

                        // model_buffer[best] = (c_k * v_k.x + a * v_g.x) / (c_k + a);
                        // model_buffer[best] = (c_k * v_k.y + a * v_g.y) / (c_k + a);
                        // model_buffer[best] = (c_k * v_k.z + a * v_g.z) / (c_k + a);
                        // model_buffer[best] = c_k + a;

                        // float4 vNormRad0 = make_float4((c_k * nRad.x + a * vNormRad.x)/ (c_k + a), (c_k * nRad.y + a * vNormRad.y)/ (c_k + a), (c_k * nRad.z + a * vNormRad.z)/ (c_k + a), (c_k * normRad.w+ a * vNormRad.w)/ (c_k + a));
                        // float3 normnrad = normalized(make_float3(vNormRad0.x,vNormRad0.y,vNormRad0.z));
                        // model_buffer[best] = normnrad.x;
                        // model_buffer[best] = normnrad.y;
                        // model_buffer[best] = normnrad.z;
                        
                    }
                } 
                else
                {
                        // model_buffer[fixed_count+i] = vPosition.x;
                        // model_buffer[fixed_count+i+ rows_mb*cols_mb] = vPosition.y;
                        // model_buffer[fixed_count+i+2*rows_mb*cols_mb] = vPosition.z;
                        // model_buffer[fixed_count+i+3*rows_mb*cols_mb] = vPosition.w;

                        // model_buffer[fixed_count+i+8*rows_mb*cols_mb] = vNormRad.x;
                        // model_buffer[fixed_count+i+9*rows_mb*cols_mb] = vNormRad.y;
                        // model_buffer[fixed_count+i+10*rows_mb*cols_mb] = vNormRad.z;
                        // model_buffer[fixed_count+i+11*rows_mb*cols_mb] = vNormRad.w;
                        // atomicAdd(count, 1);

                }
            }
        }
    }
}

void normalFusionData(DeviceArray<float>& model_buffer, int* h_count, int* h_update_count, int time, DeviceArray2D<float>& depth, const CameraModel& intr, int rows, int cols, float maxDepth, const mat33& Rmat, const float3& tvec, float weighting, DeviceArray2D<float>& vmap_pi, DeviceArray2D<float>& ct_pi, DeviceArray2D<float>& nmap_pi, DeviceArray2D<unsigned int>& index_pi, DeviceArray2D<float>& neighbours_and_vert)
{
    dim3 block (32, 8);
    dim3 grid (1, 1, 1);
    grid.x = getGridDim (depth.cols(), block.x);
    grid.y = getGridDim (depth.rows(), block.y);

    float fx = intr.fx, cx = intr.cx;
    float fy = intr.fy, cy = intr.cy;

    int *d_count, fixed_count, *d_update_count;
    fixed_count = *h_count;
    hipMalloc((void**)&d_count, sizeof(int));
    hipMalloc((void**)&d_update_count, sizeof(int));
    hipMemcpy(d_count, h_count, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_update_count, h_update_count, sizeof(int), hipMemcpyHostToDevice);

    normalFusionDataKernel<<<grid, block>>>(model_buffer, d_count, fixed_count, d_update_count, time, depth, fx, fy, cx, cy, rows, cols, maxDepth, Rmat, tvec, weighting, vmap_pi, ct_pi, nmap_pi, index_pi, neighbours_and_vert);
    hipDeviceSynchronize();
    hipMemcpy(h_count, d_count, sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(h_update_count, d_update_count, sizeof(int), hipMemcpyDeviceToHost);
    cudaSafeCall(hipGetLastError());

}
__global__ void expKernel(const PtrStepSz<float> depth, const PtrStepSz<float> vmap_pi, PtrStepSz<float> outframe, PtrStepSz<float> piframe, float fx, float fy, float cx, float cy, int rows, int cols, float maxDepth)
{
    
    int u = 120;
    int v = 160;

    // int i = v*cols + u;

    int windowsize = 8;
    int window_multiplier = 2;

    int outi, outj;

    outi=0;
    outj=0;


    for (int ui = u - windowsize*window_multiplier; ui < u + windowsize*window_multiplier; ui++)
    {
        outj = 0;
        for (int vj = v - windowsize*window_multiplier; vj < v + windowsize*window_multiplier; vj++)
        {
            float3 vPosLocal = make_float3(0, 0, 0);
            if(ui < depth.cols && ui > 0 && vj < depth.rows && vj > 0)
            {
                float z = depth.ptr(vj)[ui] /*/ 1000.f*/; // load and convert: mm -> meters

                if(z != 0 && z < maxDepth /*&& m == maskID*/) //FIXME
                {  

                    vPosLocal = make_float3(z * (ui - cx) / fx,  z * (vj - cy) / fy,  z);

                }
            }
            outframe.ptr(outj)[outi] = vPosLocal.x;
            outframe.ptr(outj + 2*windowsize*window_multiplier)[outi] = vPosLocal.y; 
            outframe.ptr(outj + 2*2*windowsize*window_multiplier)[outi] = vPosLocal.z; 

            piframe.ptr(outj)[outi] = vmap_pi.ptr(vj)[ui];
            piframe.ptr(outj + 2*windowsize*window_multiplier)[outi] = vmap_pi.ptr(vj + rows)[ui];
            piframe.ptr(outj + 2*2*windowsize*window_multiplier)[outi] = vmap_pi.ptr(vj + 2*rows)[ui];

            outj += 1;
        }
        outi +=1;
    }

}

void exp(DeviceArray2D<float>& depth, DeviceArray2D<float>& vmap_pi, DeviceArray2D<float>& outframe, DeviceArray2D<float>& piframe, const CameraModel& intr, int rows, int cols, float maxDepth)
{
    dim3 block (1, 1);
    dim3 grid (1, 1, 1);
    grid.x = getGridDim (1, block.x);
    grid.y = getGridDim (1, block.y);

    float fx = intr.fx, cx = intr.cx;
    float fy = intr.fy, cy = intr.cy;

    expKernel<<<grid, block>>>(depth, vmap_pi, outframe, piframe, fx, fy, cx, cy, rows, cols, maxDepth);
    cudaSafeCall(hipGetLastError());

}

__global__ void extractVmapKernel(float* model_buffer, int count, const mat33 Rmat_inv, const float3 tvec_inv, float* vmap_mb)
{
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    int rows_mb, cols_mb;
    rows_mb = cols_mb = 3072;

    float3 p = make_float3(model_buffer[i], model_buffer[i + rows_mb*cols_mb] ,model_buffer[i + 2*rows_mb*cols_mb]);
    float3 p_t = make_float3(0,0,0);
    // p_t = Rmat_inv * p + tvec_inv;
    vmap_mb[i*3] = p.x;
    vmap_mb[i*3 + 1] = p.y;
    vmap_mb[i*3 + 2] = p.z;
}
void extractVmap(DeviceArray<float>& model_buffer, int count, DeviceArray<float>& vmap_mb, const mat33 Rmat_inv, const float3 tvec_inv)
{
    int blocksize = 32*8;
    int numblocks = (count + blocksize - 1)/blocksize;
    vmap_mb.create(count*3);
    extractVmapKernel<<<numblocks, blocksize>>>(model_buffer, count, Rmat_inv, tvec_inv, vmap_mb);
    cudaSafeCall(hipGetLastError());
}