#include "hip/hip_runtime.h"
#include "utils.cuh"

__global__ void computeCameraVelOFKernel(float* angle_mat, float* mag_mat, float* dimg, float* ang_vel, float fx, int cols, int rows)
{

    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;


    if (x < cols && y < rows)
    {
        float Z = dimg[y*cols + x];

        if(Z != 0 /*&& Z < depthCutoff*/)
        {
            
            float theta = angle_mat[y*cols + x];
            float mag = mag_mat[y*cols + x];
            float u_ = mag * cos(theta);
            float v_ = mag * sin(theta);

            /*
            image jacobian
            [-f/Z     0    u/Z   uv/f   -(f+u^2/f)   v
               0    -f/Z   v/Z   f+v^2/f   -uv/f    -u]
            */

            float Ju_w = (x*y/f)*ang_vel[0] - (f + x*x/f)*ang_vel[1] + y*ang_vel[2];
            float Jv_w = (f + y*y/f)*ang_vel[0] - (x*y/f)*ang_vel[1] - x*ang_vel[2];

            float vx = Z * ( Ju_w - u_) / f;
            float vy = Z * ( Jv_w - v_) / f;
        }

    }

}

void computeCameraVelOF(float* angle_mat, float* mag_mat, float* dimg, float* ang_vel, float fx, int cols, int rows)
{

    dim3 block(32, 8);
    dim3 grid(1, 1, 1);
    grid.x = getGridDim(cols, block.x);
    grid.y = getGridDim(rows, block.y);

    computeCameraVelOFKernel<<<grid, block>>>(angle_mat, mag_mat, dimg, ang_vel, fx, rows, cols);
    cudaSafeCall(hipGetLastError());
}